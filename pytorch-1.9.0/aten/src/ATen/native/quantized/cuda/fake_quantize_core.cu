#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/native/quantized/fake_quant_affine.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <thrust/tuple.h>
#include <cmath>

/* Fake quantize a tensor
Args:
  output: output tensor.
  input : input tensor.
  sc:  scale to quantize the input tensor to
  zero_point: zero_point
  quant_min: minimum quantized value
  quant_max: maximum quantized value
Returns:
  Fake quantized tensor (float dtype).
*/
namespace at {
namespace native {
void fake_quantize_tensor_cachemask_kernel_cuda(
    Tensor& output,
    Tensor& mask,
    const Tensor& input,
    float scale,
    int64_t zero_point,
    int64_t quant_min,
    int64_t quant_max) {

  float inv_scale = 1.0f / scale;
  auto iter = TensorIteratorConfig()
    .check_all_same_dtype(false)
    .add_output(output)
    .add_output(mask)
    .add_input(input)
    .build();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "fake_quantize_tensor_cachemask_kernel_types", [&] {
    gpu_kernel_multiple_outputs(
      iter,
      [=] GPU_LAMBDA (scalar_t input_val) -> thrust::tuple<scalar_t, bool> {
        const auto qval = static_cast<int64_t>(std::nearbyint(input_val * inv_scale) + zero_point);
        return {
          // fake_quantized value
          (fminf(quant_max, fmaxf(quant_min, qval)) - zero_point) * scale,
          // mask for grad
          ((quant_min <= qval) && (qval <= quant_max))
        };
      }
    );
  });
}

void _fake_quantize_grad_learnable_tensor_kernel_cuda(
    TensorIterator& iter,
    float scale,
    float inv_scale,
    int64_t zero_point,
    int64_t quant_min,
    int64_t quant_max,
    float grad_factor) {
  float dscale_small = quant_min - zero_point;
  float dscale_big = quant_max - zero_point;
  gpu_kernel_multiple_outputs(
    iter, [=] GPU_LAMBDA (float XInput, float dYInput) -> thrust::tuple<float, float, float> {
      float dXOutput, dZeroPointOutput, dScaleOutput;
      int64_t xq = std::nearbyint(XInput * inv_scale) + zero_point;
      dXOutput = dYInput * (xq >= quant_min && xq <= quant_max);
      float xfq = static_cast<float>((std::max(std::min(xq, quant_max), quant_min) - zero_point) * scale);
      if (xq < quant_min || xq > quant_max) {
        dZeroPointOutput = (dYInput) * (-1) * scale * grad_factor;
        dScaleOutput = ((xq < quant_min) ? (dYInput * dscale_small) : (dYInput * dscale_big)) * grad_factor;
      } else {
        dZeroPointOutput = 0;
        dScaleOutput = (dYInput) * (xfq - (XInput)) * inv_scale * grad_factor;
      }
      return {dXOutput, dScaleOutput, dZeroPointOutput};
  });
}

REGISTER_DISPATCH(fake_quant_tensor_cachemask_stub, &fake_quantize_tensor_cachemask_kernel_cuda);
REGISTER_DISPATCH(fake_quant_grad_learnable_tensor_stub, &_fake_quantize_grad_learnable_tensor_kernel_cuda);

// Fake quantize per channel

void fake_quant_per_channel_cachemask_cuda(
    TensorIterator &iter, TensorIterator &iter_mask, int64_t quant_min, int64_t quant_max) {
  // TODO(future, optional): read once, write twice.  Not done at the moment
  //   for simplicity, as we do not expect this to be a bottleneck.
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "fake_quantize_channel_cachemask_cuda_mask_type_handling", [&] {
    // write mask
    gpu_kernel(iter_mask,
      [=] GPU_LAMBDA (scalar_t input_val, float scale, int64_t zero_point) -> bool {
        float inv_scale = 1.0f / scale;
        const auto qval = static_cast<int64_t>(std::nearbyint(input_val * inv_scale) + zero_point);
        return ((quant_min <= qval) && (qval <= quant_max));
    });

    // write fake_quant
    gpu_kernel(iter,
      [=] GPU_LAMBDA (scalar_t input_val, float scale, int64_t zero_point) -> scalar_t {
        float inv_scale = 1.0f / scale;
        return (fminf(
                    quant_max,
                    fmaxf(
                        quant_min,
                        static_cast<int64_t>(
                            std::nearbyint(input_val * inv_scale) +
                            zero_point))) -
                zero_point) *
            scale;
    });
  });
}

void _fake_quantize_grad_learnable_channel_kernel_cuda(TensorIterator &iter, int64_t quant_min, int64_t quant_max, float grad_factor) {
  gpu_kernel_multiple_outputs(iter,
    [=] GPU_LAMBDA (float x_input, float dy_input, float scale_input, float zero_point_input) -> thrust::tuple<float, float, float> {
      float dx_output, dscale_output, dzero_point_output;
      float inv_scale = 1.0f / scale_input;
      float dscale_small = quant_min - zero_point_input;
      float dscale_big = quant_max - zero_point_input;
      // Calculate gradients for X.
      int64_t xqi = std::nearbyint(x_input * inv_scale) + static_cast<int64_t>(zero_point_input);
      dx_output = dy_input * (xqi >= quant_min && xqi <= quant_max);
      // Calculate gradients for scale and zero point.
      float xfqi = static_cast<float>((std::max(std::min(xqi, quant_max), quant_min) - zero_point_input) * scale_input);
      if (xqi < quant_min || xqi > quant_max) {
        dzero_point_output = dy_input * (-1) * scale_input * grad_factor;
        dscale_output = ((xqi < quant_min) ? (dy_input * dscale_small) : (dy_input * dscale_big)) * grad_factor;
      } else {
        dzero_point_output = 0;
        dscale_output = dy_input * (xfqi - x_input) * inv_scale * grad_factor;
      }
      return {dx_output, dscale_output, dzero_point_output};
    });
}

REGISTER_DISPATCH(fake_quant_per_channel_cachemask_stub, &fake_quant_per_channel_cachemask_cuda);
REGISTER_DISPATCH(fake_quant_grad_learnable_channel_stub, &_fake_quantize_grad_learnable_channel_kernel_cuda);

} // namespace native
} // namespace at
