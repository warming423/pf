#include "hip/hip_runtime.h"
#include <ATen/native/UnaryOps.h>

#include <limits>

#include <ATen/AccumulateType.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/Math.h>
#include <ATen/native/TensorFactories.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/Math.cuh>
#include <ATen/NumericUtils.h>
#include <c10/cuda/CUDAMathCompat.h>
#include <ATen/NumericUtils.h>
#include <c10/util/complex.h>

namespace at {
namespace native {

void exp2_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      ScalarType::Half, ScalarType::BFloat16,
      iter.common_dtype(), "exp2_cuda",
      [&]() {
        gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t {
          return ::exp2(a);
        });
      });
}

void i0_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.dtype(), "i0_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return calc_i0(a);
    });
  });
}

void i0e_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.common_dtype(), "i0e_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return calc_i0e(a);
    });
  });
}

void sigmoid_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.common_dtype(), "sigmoid_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return static_cast<scalar_t>(1) / (static_cast<scalar_t>(1) + std::exp(-a));
    });
  });
}

void sinc_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(
      ScalarType::Half, ScalarType::BFloat16,
      iter.common_dtype(), "sinc_cuda",
      [&]() {
        gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
          if (a == scalar_t(0)) {
            return scalar_t(1);
          } else {
            // NVCC says constexpr var is not accessible from device
            scalar_t product = c10::detail::pi<scalar_t>() * a;
            return std::sin(product) / product;
          }
        });
      });
}

void logit_kernel_cuda(TensorIteratorBase& iter, const Scalar& eps_scalar) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      iter.common_dtype(),
      "logit_cuda",
      [&]() {
        using T_ACC = acc_type<scalar_t, true>;
        const T_ACC eps = eps_scalar.to<T_ACC>();
        if (eps < T_ACC(0)) {
          gpu_kernel(iter, [] GPU_LAMBDA(scalar_t x) -> scalar_t {
            const T_ACC x_acc = static_cast<T_ACC>(x);
            return c10::cuda::compat::log(x_acc / (T_ACC(1) - x_acc));
          });
        } else {
          const T_ACC lo = eps;
          const T_ACC hi = T_ACC(1) - eps;
          gpu_kernel(
              iter, [lo, hi] GPU_LAMBDA(scalar_t x) -> scalar_t {
                const T_ACC x_acc = static_cast<T_ACC>(x);
                T_ACC z = x_acc < lo ? lo : (x_acc > hi ? hi : x_acc);
                return c10::cuda::compat::log(z / (T_ACC(1) - z));
              });
        }
      });
}

void erf_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.common_dtype(), "erf_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::erf(a);
    });
  });
}

void erfc_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      ScalarType::Half, ScalarType::BFloat16,
      iter.common_dtype(), "erfc_cuda",
      [&]() {
        gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
          return ::erfc(a);
        });
      });
}

void erfinv_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "erfinv_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::erfinv(a);
    });
  });
}

void kaiser_window_kernel_cuda(TensorIteratorBase& iter, int64_t window_length, double beta_){
  AT_DISPATCH_FLOATING_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.dtype(), "kaiser_window_cuda", [&](){
    using T_ACC = acc_type<scalar_t, true>;
    const T_ACC inv_alpha = static_cast<T_ACC>(2.0 / (window_length - 1));
    const T_ACC beta = static_cast<T_ACC>(beta_);
    const T_ACC inv_i0_beta = 1.0 / calc_i0(beta);
    gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t a) -> scalar_t {
      T_ACC x = static_cast<T_ACC>(a) * inv_alpha - 1;
      T_ACC y = std::max<T_ACC>(0, 1 - x * x);
      return calc_i0(beta * ::sqrt(y)) * inv_i0_beta;
    });
  });
}

void entr_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      ScalarType::Half,
      ScalarType::BFloat16,
      iter.common_dtype(),
      "entr_cuda",
      [&]() {
        gpu_kernel(iter, [=] GPU_LAMBDA(scalar_t x) -> scalar_t {
          if (at::_isnan(x)) {
            return x;
          } else if (x > 0) {
            return -x * std::log(x);
          } else if (x == 0) {
            return 0;
          }
          return static_cast<scalar_t>(-INFINITY);
        });
      });
}

REGISTER_DISPATCH(exp2_stub, &exp2_kernel_cuda);
REGISTER_DISPATCH(i0_stub, &i0_kernel_cuda);
REGISTER_DISPATCH(special_i0e_stub, &i0e_kernel_cuda);
REGISTER_DISPATCH(sigmoid_stub, &sigmoid_kernel_cuda);
REGISTER_DISPATCH(sinc_stub, &sinc_kernel_cuda);
REGISTER_DISPATCH(logit_stub, &logit_kernel_cuda);
REGISTER_DISPATCH(erf_stub, &erf_kernel_cuda);
REGISTER_DISPATCH(erfc_stub, &erfc_kernel_cuda);
REGISTER_DISPATCH(erfinv_stub, &erfinv_kernel_cuda);
REGISTER_DISPATCH(kaiser_window_stub, &kaiser_window_kernel_cuda);
REGISTER_DISPATCH(special_entr_stub, &entr_kernel_cuda);

} // namespace native
} // namespace at
