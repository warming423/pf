#include "hip/hip_runtime.h"
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/Pow.h>

namespace at { namespace native {

// Forward declare some unary kernels
void rsqrt_kernel_cuda(TensorIteratorBase& iter);
void sqrt_kernel_cuda(TensorIteratorBase& iter);
void reciprocal_kernel_cuda(TensorIteratorBase& iter);

namespace {


// SFINAE doesn't work well with NVCC under Windows for math functions like pow and sqrt.
// So we need to define the functions with the explicit function signatures.
// As for pow, the following signatures are defined as the device function:
//   pow(float, int)
//   pow(double, int)
//   pow(float, float)
//   pow(double, double)
// As for sqrt, the following signatures are defined as the device function:
//   sqrt(float)
//   sqrt(double)
// As for inverse sqrt, we must define it explicitly in MSVC, otherwise the static cast will be
// applied to the result of the inline function, and thus the result is incorrect.
//   e.g. if we use 1.0 / sqrt(2) for 2 ^ (-0.5) in MSVC, we get
//          int(2 ^ (-0.5)) = int(1.0 / sqrt(2)) = int(1.0 / int(1.414)) = int(1.0 / 1) = 1
//        However, the correct result is
//          int(2 ^ (-0.5)) = int(1.0 / 1.414) = 0
#ifdef _MSC_VER
// Functions for pow
// pow for at::Half
static inline __host__ __device__ at::Half pow_(at::Half base, at::Half exp) {
  return static_cast<at::Half>(std::pow(static_cast<float>(base), static_cast<float>(exp)));
}
// pow for at::BFloat16
static inline __host__ __device__ at::BFloat16 pow_(at::BFloat16 base, at::BFloat16 exp) {
  return static_cast<at::BFloat16>(std::pow(static_cast<float>(base), static_cast<float>(exp)));
}
// pow (floating, floating/int)
template <typename Base_type, typename Exp_type>
static inline __host__ __device__ typename std::enable_if<std::is_floating_point<Base_type>::value && (std::is_same<Base_type, Exp_type>::value || std::is_same<Exp_type, int>::value), Base_type>::type
  pow_(Base_type base, Exp_type exp) {
  return std::pow(base, exp);
}
// pow (integral, integral)
template <typename Base_type, typename Exp_type>
static inline __host__ __device__ typename std::enable_if<std::is_integral<Base_type>::value && std::is_same<Base_type, Exp_type>::value, Base_type>::type
  pow_(Base_type base, Exp_type exp) {
  return native::powi(base, exp);
}
// pow (Otherwise)
template <typename Base_type, typename Exp_type>
static inline __host__ __device__ typename std::enable_if<!std::is_same<Base_type, Exp_type>::value && !std::is_same<Exp_type, int>::value, Base_type>::type
  pow_(Base_type base, Exp_type exp) {
  return static_cast<Base_type>(std::pow(static_cast<double>(base), static_cast<double>(exp)));
}
// pow (Complex)
template<typename B, typename E>
static inline __host__ __device__ B complex_pow_(B base, E exp) {
  return std::pow(base, exp);
}
#else
template <typename Base_type, typename Exp_type>
static inline __host__ __device__ Base_type pow_(Base_type base, Exp_type exp) {
  return ::pow(base, exp);
}
// pow (Otherwise)
template<typename B, typename E>
static inline __host__ __device__ B complex_pow_(B base, E exp) {
  return std::pow(base, exp);
}
#endif

void pow_tensor_tensor_kernel(TensorIteratorBase& iter) {
  if (isComplexType(iter.common_dtype())) {
    AT_DISPATCH_COMPLEX_TYPES(iter.common_dtype(), "pow_cuda", [&]() {
      gpu_kernel_with_scalars(iter, [=]GPU_LAMBDA(scalar_t base, scalar_t exp) -> scalar_t {
        return complex_pow_(base, exp);
      });
    });
  } else if (isFloatingType(iter.common_dtype())) {
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.common_dtype(), "pow_cuda", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t base, scalar_t exp) -> scalar_t {
        return pow_(base, exp);
      });
    });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(iter.common_dtype(), "pow_cuda", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t base, scalar_t exp) -> scalar_t {
        return native::powi(base, exp);
      });
    });
  }
}


template<typename Base_type, typename Exp_type>
void pow_tensor_scalar_kernel_impl(TensorIteratorBase& iter,
                                                 Exp_type exp) {
  const auto d_exp = static_cast<double>(exp);
  // .5 (sqrt), -.5 (rsqrt) and -1 (reciprocal) specializations are handled
  // in pow_tensor_scalar_kernel
  if (d_exp == 2) {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return base * base;
    });
  } else if (d_exp == 3) {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return base * base * base;
    });
  } else if (d_exp == -2) {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return 1.0 / (base * base);
    });
  } else {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return pow_(base, exp);
    });
  }
}

void pow_tensor_scalar_kernel(TensorIteratorBase& iter, const Scalar& exp_scalar) {
  // Dispatch to fast specialization for sqrt, rsqrt and reciprocal
  if (!exp_scalar.isComplex()) {
    if (exp_scalar.equal(.5)) {
      return sqrt_kernel_cuda(iter);
    } else if (exp_scalar.equal(-0.5)) {
      return rsqrt_kernel_cuda(iter);
    } else if (exp_scalar.equal(-1.0)) {
      return reciprocal_kernel_cuda(iter);
    }
  }
  if (isComplexType(iter.common_dtype()) || exp_scalar.isComplex()) {
    AT_DISPATCH_COMPLEX_TYPES(iter.common_dtype(), "pow_cuda", [&]() {
      const auto exp = exp_scalar.to<scalar_t>();
      gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t base) -> scalar_t {
        return complex_pow_(base, exp);
      });
    });
  } else if (isFloatingType(iter.common_dtype()) || exp_scalar.isIntegral(false)) {
    AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBFloat16, iter.common_dtype(), "pow_cuda", [&]() {
      const auto exp = exp_scalar.to<scalar_t>();
      pow_tensor_scalar_kernel_impl<scalar_t>(iter, exp);
    });
  } else {
    const auto exp = exp_scalar.to<float>();
    AT_DISPATCH_INTEGRAL_TYPES(iter.common_dtype(), "pow_cuda", [&]() {
      pow_tensor_scalar_kernel_impl<scalar_t>(iter, exp);
    });
  }
}

} // anonymous namespace

REGISTER_DISPATCH(pow_tensor_tensor_stub, &pow_tensor_tensor_kernel);
REGISTER_DISPATCH(pow_tensor_scalar_stub, &pow_tensor_scalar_kernel);

}} // namespace at::native
