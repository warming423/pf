#include "hip/hip_runtime.h"
#include <ATen/Context.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/Dispatch.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/PinnedMemoryAllocator.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/CUDASolver.h>
#include <ATen/cuda/CUDABlas.h>
#include <ATen/cuda/CUDAEvent.h>
#include <c10/cuda/CUDAStream.h>

#include <ATen/native/LinearAlgebraUtils.h>
#include <ATen/native/cuda/MiscUtils.h>
#include <ATen/native/cuda/BatchLinearAlgebraLib.h>

namespace at {
namespace native {

// Some cuBLAS and cuSOLVER batched routines require input to be a device array of pointers to device individual matrices
// 'input' must be a contiguous tensor
template <typename scalar_t>
static Tensor get_device_pointers(const Tensor& input) {
  auto input_data = input.data_ptr<scalar_t>();
  int64_t input_mat_stride = matrixStride(input);

  // cublas/cusolver interface requires 'int'
  int batch_size = cuda_int_cast(batchCount(input), "batch_size");

  // if batch_size==0, then start=0 and end=0
  // if input_mat_stride==0, then step=sizeof(scalar_t)
  return at::arange(
      /*start=*/reinterpret_cast<int64_t>(input_data),
      /*end=*/reinterpret_cast<int64_t>(input_data + batch_size * input_mat_stride),
      /*step=*/static_cast<int64_t>(std::max<int64_t>(input_mat_stride, 1) * sizeof(scalar_t)),
      input.options().dtype(at::kLong));
}

template <typename scalar_t>
void apply_geqrf_batched(const Tensor& input, const Tensor& tau) {
// AMD ROCm backend is implemented via rewriting all CUDA calls to HIP
// rocBLAS does not implement BLAS-like extensions of cuBLAS, they're in rocSOLVER
// rocSOLVER is currently not used in ATen, therefore we raise an error in this case
#ifndef CUDART_VERSION
  TORCH_CHECK(false, "geqrf: Batched version is supported only with cuBLAS backend.")
#else
  auto batch_size = cuda_int_cast(batchCount(input), "batch_size");
  auto m = cuda_int_cast(input.size(-2), "m");
  auto n = cuda_int_cast(input.size(-1), "n");
  auto lda = std::max<int>(1, m);

  // cuBLAS batched geqrf requires input to be the device array of pointers to device single matrices
  Tensor input_ptr_array = get_device_pointers<scalar_t>(input);
  Tensor tau_ptr_array = get_device_pointers<scalar_t>(tau.unsqueeze(-1));
  auto input_ptr_array_data = reinterpret_cast<scalar_t**>(input_ptr_array.data_ptr());
  auto tau_ptr_array_data = reinterpret_cast<scalar_t**>(tau_ptr_array.data_ptr());

  int info;
  auto handle = at::cuda::getCurrentCUDABlasHandle();
  at::cuda::blas::geqrfBatched(handle, m, n, input_ptr_array_data, lda, tau_ptr_array_data, &info, batch_size);

  // info only indicates wrong arguments to geqrfBatched call
  // info is a host variable, we can check it without device synchronization
  TORCH_INTERNAL_ASSERT(info == 0);
#endif
}

void geqrf_batched_cublas(const Tensor& input, const Tensor& tau) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(input.scalar_type(), "geqrf_batched_cuda", [&]{
    apply_geqrf_batched<scalar_t>(input, tau);
  });
}

template <typename scalar_t>
static void apply_triangular_solve(Tensor& A, Tensor& B, bool upper, bool transpose, bool conjugate_transpose, bool unitriangular) {
  hipblasFillMode_t uplo = upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER;
  hipblasOperation_t trans = transpose ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  trans = conjugate_transpose ? HIPBLAS_OP_C : trans;
  hipblasDiagType_t diag = unitriangular ? HIPBLAS_DIAG_UNIT : HIPBLAS_DIAG_NON_UNIT;
  hipblasSideMode_t side = HIPBLAS_SIDE_LEFT;

  auto A_data = A.data_ptr<scalar_t>();
  auto B_data = B.data_ptr<scalar_t>();
  auto A_mat_stride = matrixStride(A);
  auto B_mat_stride = matrixStride(B);
  auto batch_size = batchCount(A);
  auto m = cuda_int_cast(A.size(-2), "m");
  auto n = cuda_int_cast(A.size(-1), "n");
  auto nrhs = cuda_int_cast(B.size(-1), "nrhs");
  auto lda = std::max<int>(1, m);

  auto alpha = scalar_t{1};

  for (decltype(batch_size) i = 0; i < batch_size; i++) {
    scalar_t* A_working_ptr = &A_data[i * A_mat_stride];
    scalar_t* B_working_ptr = &B_data[i * B_mat_stride];
    auto handle = at::cuda::getCurrentCUDABlasHandle();
    at::cuda::blas::trsm(handle, side, uplo, trans, diag, n, nrhs, &alpha, A_working_ptr, lda, B_working_ptr, lda);
  }
}

void triangular_solve_cublas(Tensor& A, Tensor& B, Tensor& infos, bool upper, bool transpose, bool conjugate_transpose, bool unitriangular) {
  (void)infos; // unused
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(A.scalar_type(), "triangular_solve_cuda", [&]{
    apply_triangular_solve<scalar_t>(A, B, upper, transpose, conjugate_transpose, unitriangular);
  });
}

template <typename scalar_t>
static void apply_triangular_solve_batched(Tensor& A, Tensor& B, bool upper, bool transpose, bool conjugate_transpose, bool unitriangular) {
  hipblasFillMode_t uplo = upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER;
  hipblasOperation_t trans = transpose ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  trans = conjugate_transpose ? HIPBLAS_OP_C : trans;
  hipblasDiagType_t diag = unitriangular ? HIPBLAS_DIAG_UNIT : HIPBLAS_DIAG_NON_UNIT;
  hipblasSideMode_t side = HIPBLAS_SIDE_LEFT;

  auto A_data = A.data_ptr<scalar_t>();
  auto B_data = B.data_ptr<scalar_t>();
  auto A_mat_stride = matrixStride(A);
  auto B_mat_stride = matrixStride(B);
  auto batch_size = cuda_int_cast(batchCount(A), "batch_size");
  auto m = cuda_int_cast(A.size(-2), "m");
  auto n = cuda_int_cast(A.size(-1), "n");
  auto nrhs = cuda_int_cast(B.size(-1), "nrhs");
  auto lda = std::max<int>(1, m);

  auto alpha = scalar_t{1};

  // cuBLAS batched trsm requires input to be the device array of pointers to device single matrices
  Tensor A_ptr_array = get_device_pointers<scalar_t>(A);
  Tensor B_ptr_array = get_device_pointers<scalar_t>(B);
  auto A_ptr_array_data = reinterpret_cast<scalar_t**>(A_ptr_array.data_ptr());
  auto B_ptr_array_data = reinterpret_cast<scalar_t**>(B_ptr_array.data_ptr());

  auto handle = at::cuda::getCurrentCUDABlasHandle();
  at::cuda::blas::trsmBatched(handle, side, uplo, trans, diag, n, nrhs, &alpha, A_ptr_array_data, lda, B_ptr_array_data, lda, batch_size);
}

void triangular_solve_batched_cublas(Tensor& A, Tensor& B, Tensor& infos, bool upper, bool transpose, bool conjugate_transpose, bool unitriangular) {
  (void)infos; // unused
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(A.scalar_type(), "triangular_solve_cuda", [&]{
    apply_triangular_solve_batched<scalar_t>(A, B, upper, transpose, conjugate_transpose, unitriangular);
  });
}

template <typename scalar_t>
inline void apply_gels_batched(const Tensor& A, Tensor& B, Tensor& infos) {
// AMD ROCm backend is implemented via rewriting all CUDA calls to HIP
// rocBLAS does not implement BLAS-like extensions of cuBLAS, they're in rocSOLVER
// rocSOLVER is currently not used in ATen, therefore we raise an error in this case
#ifndef CUDART_VERSION
  TORCH_CHECK(false, "torch.linalg.lstsq: Batched version is supported only with cuBLAS backend.")
#else
  auto trans = HIPBLAS_OP_N;
  auto m = cuda_int_cast(A.size(-2), "m");
  auto n = cuda_int_cast(A.size(-1), "n");

  auto nrhs = cuda_int_cast(B.size(-1), "nrhs");
  // cuBLAS from cuda10 and older doesn't work with nrhs == 0 (cuda11 works)
  // so we need to put this early return
  if (nrhs == 0) {
    return;
  }

  auto batch_size = cuda_int_cast(batchCount(B), "batch_size");
  auto lda = std::max<int>(1, m);
  auto ldb = std::max<int>(1, m);

  // cuBLAS's requirement
  TORCH_CHECK(
    m >= n,
    "torch.linalg.lstsq: only overdetermined systems (input.size(-2) >= input.size(-1)) are allowed on CUDA with cuBLAS backend.");

  // cuBLAS documentation says:
  // Matrices Aarray[i] should not overlap; otherwise, undefined behavior is expected.
  // explicitly broadcast the batch dimensions of A
  IntArrayRef A_batch_sizes(A.sizes().data(), A.dim() - 2);
  IntArrayRef B_batch_sizes(B.sizes().data(), B.dim() - 2);
  std::vector<int64_t> expand_batch_portion = at::infer_size(A_batch_sizes, B_batch_sizes);
  expand_batch_portion.insert(expand_batch_portion.end(), {A.size(-2), A.size(-1)});
  Tensor A_expanded = A.expand({expand_batch_portion});
  Tensor A_broadcasted = cloneBatchedColumnMajor(A_expanded);

  // cuBLAS batched gels requires input to be the device array of pointers to device single matrices
  Tensor A_ptr_array = get_device_pointers<scalar_t>(A_broadcasted);
  Tensor B_ptr_array = get_device_pointers<scalar_t>(B);
  auto A_ptr_array_data = reinterpret_cast<scalar_t**>(A_ptr_array.data_ptr());
  auto B_ptr_array_data = reinterpret_cast<scalar_t**>(B_ptr_array.data_ptr());

  auto infos_data = infos.data_ptr<int>();
  auto handle = at::cuda::getCurrentCUDABlasHandle();
  int info;

  at::cuda::blas::gelsBatched<scalar_t>(
    handle, trans, m, n, nrhs,
    A_ptr_array_data, lda,
    B_ptr_array_data, ldb,
    &info,
    infos_data,
    batch_size);

  // negative info indicates that an argument to gelsBatched call is invalid
  TORCH_INTERNAL_ASSERT(info == 0);
#endif
}

// This is a type dispatching helper function for 'apply_gels_batched'
void gels_batched_cublas(const Tensor& a, Tensor& b, Tensor& infos) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(a.scalar_type(), "gels_batched_cublas", [&]{
    apply_gels_batched<scalar_t>(a, b, infos);
  });
}

#ifdef USE_CUSOLVER

inline static Tensor column_major_identity_matrix_like(const Tensor& self) {
  auto size = self.sizes();
  auto size_slice = IntArrayRef(size.data(), size.size()-1);
  return at::ones(size_slice, self.options()).diag_embed().transpose(-2, -1);
}

template <typename scalar_t>
inline static void _apply_single_inverse_helper(scalar_t* self_ptr, scalar_t* self_inv_ptr, int* ipiv_ptr, int* info_getrf_ptr, int* info_getrs_ptr, int n, int lda) {
  // self_inv_ptr should already be an identity matrix

  auto handle = at::cuda::getCurrentCUDASolverDnHandle();
  at::cuda::solver::getrf<scalar_t>(handle, n, n, self_ptr, lda, ipiv_ptr, info_getrf_ptr);
  at::cuda::solver::getrs<scalar_t>(handle, n, n, self_ptr, lda, ipiv_ptr, self_inv_ptr, lda, info_getrs_ptr);
}

template <typename scalar_t>
static void apply_batched_inverse_lib(Tensor& self, Tensor& self_inv, Tensor& infos_getrf, Tensor& infos_getrs) {
  const int batch_size = cuda_int_cast(batchCount(self), "batchCount");
  const int n = cuda_int_cast(self.size(-2), "self.size(-2)");
  const int lda = std::max<int>(1, n);

  auto self_data = self.data_ptr<scalar_t>();
  auto self_mat_stride = matrixStride(self);
  auto self_inv_data = self_inv.data_ptr<scalar_t>();
  auto self_inv_mat_stride = matrixStride(self_inv);

  auto infos_getrf_data = infos_getrf.data_ptr<int>();
  auto infos_getrs_data = infos_getrs.data_ptr<int>();

  auto& allocator = *::c10::cuda::CUDACachingAllocator::get();

  // Heuristic: For small batch size or large matrix size, we use for-loop to iterate over the batches instead of
  //            calling the batched cublas routine.
  if (batch_size <= 8 || /* batch_size > 8 && */ n >= 512) {
    for (int64_t i = 0; i < batch_size; i++) {
      auto dataPtr = allocator.allocate(sizeof(int) * lda);
      int* pivot = reinterpret_cast<int*>(dataPtr.get());

      int* infos_getrf_working_ptr = &infos_getrf_data[i];
      int* infos_getrs_working_ptr = &infos_getrs_data[i];

      _apply_single_inverse_helper<scalar_t>(
        &self_data[i * self_mat_stride], &self_inv_data[i * self_inv_mat_stride], pivot, infos_getrf_working_ptr, infos_getrs_working_ptr, n, lda);
    }
  } else {
    // cublas batched kernels require input be "device array of device pointers"
    Tensor self_array = at::arange(
      reinterpret_cast<int64_t>(self_data),
      reinterpret_cast<int64_t>(&self_data[(batch_size-1) * self_mat_stride]) + 1,
      static_cast<int64_t>(self_mat_stride * sizeof(scalar_t)), self.options().dtype(at::kLong));
    Tensor self_inv_array = at::arange(
      reinterpret_cast<int64_t>(self_inv_data),
      reinterpret_cast<int64_t>(&self_inv_data[(batch_size-1) * self_inv_mat_stride]) + 1,
      static_cast<int64_t>(self_inv_mat_stride * sizeof(scalar_t)), self.options().dtype(at::kLong));

    auto dataPtr = allocator.allocate(sizeof(int)*batch_size*lda);
    int* ipiv_array = reinterpret_cast<int*>(dataPtr.get());

    at::cuda::blas::getrfBatched<scalar_t>(n, reinterpret_cast<scalar_t**>(self_array.data_ptr()), lda,
      ipiv_array, infos_getrf_data, batch_size);

    at::cuda::blas::getriBatched<scalar_t>(n, reinterpret_cast<scalar_t**>(self_array.data_ptr()), lda,
      ipiv_array, reinterpret_cast<scalar_t**>(self_inv_array.data_ptr()), lda, infos_getrs_data, batch_size);
  }
}

template <typename scalar_t>
static void apply_single_inverse_lib(const Tensor& self, Tensor& self_inv, Tensor& infos_getrf, Tensor& infos_getrs) {
  int n = cuda_int_cast(self.size(-2), "self.size(-2)");
  int lda = std::max<int>(1, n);

  Tensor ipiv = at::empty({lda}, self.options().dtype(at::kInt));

  _apply_single_inverse_helper<scalar_t>(
    self.data_ptr<scalar_t>(), self_inv.data_ptr<scalar_t>(), ipiv.data_ptr<int>(), infos_getrf.data_ptr<int>(), infos_getrs.data_ptr<int>(), n, lda);
}

// This is a type dispatching helper function for 'apply_batched_inverse_lib' and 'apply_single_inverse_lib'
Tensor& _linalg_inv_out_helper_cuda_lib(Tensor& result, Tensor& infos_getrf, Tensor& infos_getrs) {
  // assuming result is in column major order and contains the matrices to invert
  Tensor input_working_copy = cloneBatchedColumnMajor(result);

  // for getrf + getrs (cusolver path)
  // result should be filled with identity matrices
  result.zero_();
  result.diagonal(/*offset=*/0, /*dim1=*/-2, /*dim2=*/-1).fill_(1);

  const int batch_size = cuda_int_cast(batchCount(result), "batchCount");

  if (result.dim() > 2) {
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(result.scalar_type(), "linalg_inv_out_cuda", [&]{
      apply_batched_inverse_lib<scalar_t>(
        input_working_copy, result, infos_getrf, infos_getrs);
    });
  } else {
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(result.scalar_type(), "linalg_inv_out_cuda", [&]{
      apply_single_inverse_lib<scalar_t>(input_working_copy, result, infos_getrf, infos_getrs);
    });
  }

  return result;
}

// entrance of calculations of `inverse` using cusolver getrf + getrs, cublas getrfBatched + getriBatched
Tensor _inverse_helper_cuda_lib(const Tensor& self) {
  Tensor self_working_copy = cloneBatchedColumnMajor(self);
  Tensor self_inv_working_copy = column_major_identity_matrix_like(self_working_copy);
  const int batch_size = cuda_int_cast(batchCount(self), "batchCount");

  if (self.dim() > 2 && batch_size > 1) {
    Tensor infos_getrf = at::zeros({std::max<int64_t>(1, batchCount(self))}, self.options().dtype(kInt));
    Tensor infos_getrs = at::zeros({std::max<int64_t>(1, batchCount(self))}, self.options().dtype(kInt));
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(self.scalar_type(), "inverse_cuda", [&]{
      apply_batched_inverse_lib<scalar_t>(
        self_working_copy, self_inv_working_copy, infos_getrf, infos_getrs);
    });
    batchCheckErrors(infos_getrf, "inverse_cuda");
    batchCheckErrors(infos_getrs, "inverse_cuda");
  } else {
    Tensor infos_getrf = at::zeros({1}, self.options().dtype(kInt));
    Tensor infos_getrs = at::zeros({1}, self.options().dtype(kInt));
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(self.scalar_type(), "inverse_cuda", [&]{
      apply_single_inverse_lib<scalar_t>(self_working_copy, self_inv_working_copy, infos_getrf, infos_getrs);
    });
    batchCheckErrors(infos_getrf, "inverse_cuda");
    batchCheckErrors(infos_getrs, "inverse_cuda");
  }

  return self_inv_working_copy;
}

// call cusolver gesvdj function to calculate svd
template<typename scalar_t>
inline static void _apply_svd_lib_gesvdj(const Tensor& self, Tensor& U, Tensor& S, Tensor& VT, Tensor& infos, bool compute_uv, bool some) {
  using value_t = typename c10::scalar_value_type<scalar_t>::type;
  auto self_data = self.data_ptr<scalar_t>();
  auto U_data = U.data_ptr<scalar_t>();
  auto S_data = S.data_ptr<value_t>();
  auto VT_data = VT.data_ptr<scalar_t>();
  auto self_stride = matrixStride(self);
  auto U_stride = matrixStride(U);
  auto S_stride = S.size(-1);
  auto VT_stride = matrixStride(VT);

  int batchsize = cuda_int_cast(batchCount(self), "batch size");
  int m = cuda_int_cast(self.size(-2), "m");
  int n = cuda_int_cast(self.size(-1), "n");
  int lda = std::max<int>(1, m);
  int ldvt = std::max<int>(1, n);

  for(int i = 0; i < batchsize; i++){
    // gesvdj_params controls the numerical accuracy of cusolver gesvdj iterations on GPU
    hipsolverGesvdjInfo_t gesvdj_params;
    TORCH_CUSOLVER_CHECK(hipsolverDnCreateGesvdjInfo(&gesvdj_params));
    // TORCH_CUSOLVER_CHECK(hipsolverDnXgesvdjSetTolerance(gesvdj_params, 1.0e-7));
    // TORCH_CUSOLVER_CHECK(hipsolverDnXgesvdjSetMaxSweeps(gesvdj_params, 15));

    auto handle = at::cuda::getCurrentCUDASolverDnHandle();
    auto jobz = compute_uv ? HIPSOLVER_EIG_MODE_VECTOR : HIPSOLVER_EIG_MODE_NOVECTOR;
    at::cuda::solver::gesvdj<scalar_t>(
      handle, jobz, /*econ=*/ some ? 1 : 0, m, n,
      self_data + i * self_stride,
      lda,
      S_data + i * S_stride,
      U_data + i * U_stride,
      lda,
      VT_data + i * VT_stride,
      ldvt,
      infos.data_ptr<int>() + i,
      gesvdj_params
    );

    TORCH_CUSOLVER_CHECK(hipsolverDnDestroyGesvdjInfo(gesvdj_params));
  }
}

// wrapper around _apply_svd_lib_gesvdj that handles dtype dispatch,
// creates a working copy of the input, and creates V^H from the V returned by gesvdj
inline static void apply_svd_lib_gesvdj(const Tensor& self, Tensor& U, Tensor& S, Tensor& VT, Tensor& infos, bool compute_uv, bool some) {
  const int64_t m = self.size(-2);
  const int64_t n = self.size(-1);
  Tensor self_working_copy = cloneBatchedColumnMajor(self);
  VT = VT.transpose(-2, -1);  // gesvdj returns V instead of V^H

  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(self.scalar_type(), "svd_cuda_gesvdj", [&] {
    _apply_svd_lib_gesvdj<scalar_t>(self_working_copy, U, S, VT, infos, compute_uv, some);
  });
}

// call cusolver gesvdj batched function to calculate svd
template<typename scalar_t>
inline static void _apply_svd_lib_gesvdjBatched(const Tensor& self, Tensor& U, Tensor& S, Tensor& VT, Tensor& infos, bool compute_uv) {
  using value_t = typename c10::scalar_value_type<scalar_t>::type;
  auto self_data = self.data_ptr<scalar_t>();
  auto U_data = U.data_ptr<scalar_t>();
  auto S_data = S.data_ptr<value_t>();
  auto VT_data = VT.data_ptr<scalar_t>();
  auto self_stride = matrixStride(self);
  auto U_stride = matrixStride(U);
  auto S_stride = S.size(-1);
  auto VT_stride = matrixStride(VT);

  int batchsize = cuda_int_cast(batchCount(self), "batch size");
  int m = cuda_int_cast(self.size(-2), "m");
  int n = cuda_int_cast(self.size(-1), "n");
  int lda = std::max<int>(1, m);
  int ldvt = std::max<int>(1, n);

  TORCH_INTERNAL_ASSERT(m <= 32 && n <= 32, "gesvdjBatched requires both matrix dimensions not greater than 32, but got "
                        "m = ", m, " n = ", n);

  // gesvdj_params controls the numerical accuracy of cusolver gesvdj iterations on GPU
  hipsolverGesvdjInfo_t gesvdj_params;
  TORCH_CUSOLVER_CHECK(hipsolverDnCreateGesvdjInfo(&gesvdj_params));
  // TORCH_CUSOLVER_CHECK(hipsolverDnXgesvdjSetTolerance(gesvdj_params, 1.0e-7));
  // TORCH_CUSOLVER_CHECK(hipsolverDnXgesvdjSetMaxSweeps(gesvdj_params, 15));
  TORCH_CUSOLVER_CHECK(hipsolverDnXgesvdjSetSortEig(gesvdj_params, 1));

  auto handle = at::cuda::getCurrentCUDASolverDnHandle();
  auto jobz = compute_uv ? HIPSOLVER_EIG_MODE_VECTOR : HIPSOLVER_EIG_MODE_NOVECTOR;
  at::cuda::solver::gesvdjBatched<scalar_t>(
    handle, jobz, m, n, self_data, lda, S_data, U_data, lda, VT_data, ldvt,
    infos.data_ptr<int>(), gesvdj_params, batchsize
  );

  TORCH_CUSOLVER_CHECK(hipsolverDnDestroyGesvdjInfo(gesvdj_params));
}

// wrapper around _apply_svd_lib_gesvdjBatched that handles dtype dispatch,
// creates a working copy of the input, and creates V^H from the V returned by gesvdj
inline static void apply_svd_lib_gesvdjBatched(const Tensor& self, Tensor& U, Tensor& S, Tensor& VT, Tensor& infos, bool compute_uv) {
  const int64_t m = self.size(-2);
  const int64_t n = self.size(-1);
  Tensor self_working_copy = cloneBatchedColumnMajor(self);
  VT = VT.transpose(-2, -1);  // gesvdj returns V instead of V^H

  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(self.scalar_type(), "svd_cuda_gesvdjBatched", [&] {
    _apply_svd_lib_gesvdjBatched<scalar_t>(self_working_copy, U, S, VT, infos, compute_uv);
  });
}

// entrance of calculations of `svd` using cusolver gesvdj and gesvdjBatched
std::tuple<Tensor, Tensor, Tensor> _svd_helper_cuda_lib(const Tensor& self, bool some, bool compute_uv) {
  const int64_t batch_size = batchCount(self);
  at::Tensor infos = at::zeros({batch_size}, self.options().dtype(at::kInt));
  const int64_t m = self.size(-2);
  const int64_t n = self.size(-1);
  const int64_t k = std::min(m, n);

  Tensor U_working_copy, S_working_copy, VT_working_copy;
  std::tie(U_working_copy, S_working_copy, VT_working_copy) = \
    _create_U_S_VT(self, some, compute_uv, /* svd_use_cusolver = */ true);
  // U, S, V working copies are already column majored now

  // heuristic for using `gesvdjBatched` over `gesvdj`
  if (m <= 32 && n <= 32 && batch_size > 1 && (!some || m == n)) {
    apply_svd_lib_gesvdjBatched(self, U_working_copy, S_working_copy, VT_working_copy, infos, compute_uv);
  } else {
    apply_svd_lib_gesvdj(self, U_working_copy, S_working_copy, VT_working_copy, infos, compute_uv, some);
  }

  // A device-host sync will be performed.
  batchCheckErrors(infos, "svd_cuda");

  if (!compute_uv) {
    VT_working_copy.zero_();
    U_working_copy.zero_();
  }

  if (some) {
    VT_working_copy = VT_working_copy.narrow(-2, 0, k);
  }

  // so far we have computed VT, but torch.svd returns V instead. Adjust accordingly.
  VT_working_copy.transpose_(-2, -1);
  return std::make_tuple(U_working_copy, S_working_copy, VT_working_copy);
}


// Implementation of Cholesky decomposition using looped cusolverDn<T>potrf or hipsolverDnXpotrf (64-bit)
template<typename scalar_t>
inline static void apply_cholesky_cusolver_potrf_looped(const Tensor& self_working_copy, bool upper, const Tensor& infos) {
  auto handle = at::cuda::getCurrentCUDASolverDnHandle();
  const auto uplo = upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER;
  const int64_t n = self_working_copy.size(-1);
  const int64_t lda = std::max<int64_t>(1, n);
  const int64_t batch_size = batchCount(self_working_copy);
  const int64_t matrix_stride = matrixStride(self_working_copy);

  scalar_t* self_working_copy_ptr = self_working_copy.data_ptr<scalar_t>();
  int* infos_ptr = infos.data_ptr<int>();

#ifdef USE_CUSOLVER_64_BIT
  size_t worksize_device;
  size_t worksize_host;
  hipsolverDnParams_t params;
  hipDataType datatype = at::cuda::solver::get_cusolver_datatype<scalar_t>();
  TORCH_CUSOLVER_CHECK(hipsolverDnCreateParams(&params));
  at::cuda::solver::xpotrf_buffersize(handle, params, uplo, n, datatype, nullptr, lda, datatype, &worksize_device, &worksize_host);

  // allocate workspace storage
  auto& device_allocator = *at::cuda::getCUDADeviceAllocator();
  auto workdata_device = device_allocator.allocate(worksize_device * batch_size);
  void* workdata_device_ptr = workdata_device.get();

  auto& host_allocator = *at::getCPUAllocator();
  auto workdata_host = host_allocator.allocate(worksize_host * batch_size);
  void* workdata_host_ptr = workdata_host.get();

  for (int64_t i = 0; i < batch_size; i++) {
    at::cuda::solver::xpotrf(
      handle, params, uplo, n, datatype,
      self_working_copy_ptr + i * matrix_stride,
      lda, datatype,
      (char*)workdata_device_ptr + i * worksize_device, worksize_device,
      (char*)workdata_host_ptr + i * worksize_host, worksize_host,
      infos_ptr + i
    );
  }

  TORCH_CUSOLVER_CHECK(hipsolverDnDestroyParams(params));
#else // USE_CUSOLVER_64_BIT
  int n_32 = cuda_int_cast(n, "n");
  int lda_32 = cuda_int_cast(lda, "lda");
  int lwork;
  at::cuda::solver::potrf_buffersize<scalar_t>(
    handle, uplo, n_32, nullptr, lda_32, &lwork);

   // allocate workspace storage
  auto& allocator = *at::cuda::getCUDADeviceAllocator();
  auto work_data = allocator.allocate(sizeof(scalar_t)*lwork * batch_size);
  scalar_t* work_data_ptr = static_cast<scalar_t*>(work_data.get());

  for (int64_t i = 0; i < batch_size; i++) {
    at::cuda::solver::potrf<scalar_t>(
      handle, uplo, n_32,
      self_working_copy_ptr + i * matrix_stride,
      lda_32,
      work_data_ptr + i * lwork,
      lwork,
      infos_ptr + i
    );
  }
#endif // USE_CUSOLVER_64_BIT
}

// Implementation of Cholesky decomposition using batched cusolverDn<T>potrfBatched
// Warning: cusolverDn<T>potrfBatched doesn't work quite well when matrix size or batch size is zero.
// If you write your own C++ extension and use this function, make sure you do a zero numel check for the input.
template<typename scalar_t>
inline static void apply_cholesky_cusolver_potrfBatched(const Tensor& self_working_copy, bool upper, const Tensor& infos) {
  auto handle = at::cuda::getCurrentCUDASolverDnHandle();
  const auto uplo = upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER;
  const int n = cuda_int_cast(self_working_copy.size(-1), "n");
  const int lda = std::max<int>(1, n);

  const int batch_size = cuda_int_cast(batchCount(self_working_copy), "batch_size");

  // cusolver batched kernels require input be "device array of device pointers"
  Tensor self_working_copy_array = get_device_pointers<scalar_t>(self_working_copy);

  at::cuda::solver::potrfBatched<scalar_t>(
    handle, uplo, n,
    reinterpret_cast<scalar_t**>(self_working_copy_array.data_ptr()),
    lda, infos.data_ptr<int>(), batch_size);
}

void cholesky_helper_cusolver(const Tensor& input, bool upper, const Tensor& info) {
  if (input.numel() == 0) {
    return;
  }

  if (use_cusolver_potrf_batched_ && batchCount(input) > 1) {
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(input.scalar_type(), "cholesky_cusolver", [&] {
      apply_cholesky_cusolver_potrfBatched<scalar_t>(input, upper, info);
    });
  } else {
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(input.scalar_type(), "cholesky_cusolver", [&] {
      apply_cholesky_cusolver_potrf_looped<scalar_t>(input, upper, info);
    });
  }
}


template<typename scalar_t>
inline static void apply_cholesky_cusolver_potrs(Tensor& self_working_copy, const Tensor& A_column_major_copy, bool upper, Tensor& infos) {
  auto handle = at::cuda::getCurrentCUDASolverDnHandle();
  const auto uplo = upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER;
  const int64_t n = self_working_copy.size(-2);
  const int64_t nrhs = self_working_copy.size(-1);
  const int64_t lda = std::max<int64_t>(1, n);
  const int64_t batch_size = batchCount(self_working_copy);
  const int64_t self_matrix_stride = matrixStride(self_working_copy);
  scalar_t* self_working_copy_ptr = self_working_copy.data_ptr<scalar_t>();

  const scalar_t* A_ptr = A_column_major_copy.data_ptr<scalar_t>();
  const int64_t A_matrix_stride = matrixStride(A_column_major_copy);
  const int64_t ldb = std::max<int64_t>(1, A_column_major_copy.size(-1));

  int* infos_ptr = infos.data_ptr<int>();

#ifdef USE_CUSOLVER_64_BIT
  hipsolverDnParams_t params;
  hipDataType datatype = at::cuda::solver::get_cusolver_datatype<scalar_t>();
  TORCH_CUSOLVER_CHECK(hipsolverDnCreateParams(&params));

  for (int64_t i = 0; i < batch_size; i++) {
    at::cuda::solver::xpotrs(
      handle, params, uplo, n, nrhs, datatype,
      A_ptr + i * A_matrix_stride,
      lda, datatype,
      self_working_copy_ptr + i * self_matrix_stride,
      ldb,
      infos_ptr
    );
  }

  TORCH_CUSOLVER_CHECK(hipsolverDnDestroyParams(params));
#else // USE_CUSOLVER_64_BIT
  int n_32 = cuda_int_cast(n, "n");
  int nrhs_32 = cuda_int_cast(nrhs, "nrhs");
  int lda_32 = cuda_int_cast(lda, "lda");
  int ldb_32 = cuda_int_cast(ldb, "ldb");

  for (int64_t i = 0; i < batch_size; i++) {
    at::cuda::solver::potrs<scalar_t>(
      handle, uplo, n_32, nrhs_32,
      A_ptr + i * A_matrix_stride,
      lda_32,
      self_working_copy_ptr + i * self_matrix_stride,
      ldb_32,
      infos_ptr
    );
  }
#endif // USE_CUSOLVER_64_BIT
}


// This code path is only dispatched to if MAGMA is not linked in the pytorch build.
// cusolverDn<t>potrsBatched only supports nrhs == 1
template<typename scalar_t>
inline static void apply_cholesky_cusolver_potrsBatched(Tensor& self_working_copy, const Tensor& A_column_major_copy, bool upper, Tensor& infos) {
  auto handle = at::cuda::getCurrentCUDASolverDnHandle();
  const auto uplo = upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER;
  const int64_t n = self_working_copy.size(-2);
  const int64_t nrhs = self_working_copy.size(-1);
  const int64_t lda = std::max<int64_t>(1, n);
  const int64_t batch_size = batchCount(self_working_copy);
  const int64_t self_matrix_stride = matrixStride(self_working_copy);
  scalar_t* self_working_copy_ptr = self_working_copy.data_ptr<scalar_t>();

  const scalar_t* A_ptr = A_column_major_copy.data_ptr<scalar_t>();
  const int64_t A_matrix_stride = matrixStride(A_column_major_copy);
  const int64_t ldb = std::max<int64_t>(1, A_column_major_copy.size(-1));

  int* infos_ptr = infos.data_ptr<int>();

  auto self_ptr_array = get_device_pointers<scalar_t>(self_working_copy);
  auto A_ptr_array = get_device_pointers<scalar_t>(A_column_major_copy);

  at::cuda::solver::potrsBatched(
    handle, uplo,
    cuda_int_cast(n, "n"),
    cuda_int_cast(nrhs, "nrhs"),
    reinterpret_cast<scalar_t**>(A_ptr_array.data_ptr()),
    cuda_int_cast(lda, "lda"),
    reinterpret_cast<scalar_t**>(self_ptr_array.data_ptr()),
    cuda_int_cast(ldb, "ldb"),
    infos_ptr,
    cuda_int_cast(batch_size, "batch_size")
  );
}

Tensor _cholesky_solve_helper_cuda_cusolver(const Tensor& self, const Tensor& A, bool upper) {
  const int64_t batch_size = batchCount(self);
  at::Tensor infos = at::zeros({1}, self.options().dtype(at::kInt));
  at::Tensor self_working_copy = cloneBatchedColumnMajor(self);
  at::Tensor A_column_major_copy = cloneBatchedColumnMajor(A);

  const int64_t nrhs = self_working_copy.size(-1);

  // cusolverDn<t>potrsBatched only supports nrhs == 1
  if (batch_size > 1 && nrhs == 1) {
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(self.scalar_type(), "cholesky_cuda_potrs_batched", [&] {
      apply_cholesky_cusolver_potrsBatched<scalar_t>(self_working_copy, A_column_major_copy, upper, infos);
    });
  } else {
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(self.scalar_type(), "cholesky_cuda_potrs", [&] {
      apply_cholesky_cusolver_potrs<scalar_t>(self_working_copy, A_column_major_copy, upper, infos);
    });
  }

  // info from potrs and potrsBatched only report if the i-th parameter is wrong, not about the matrix singularity, etc.
  // So we don't need to check it all the time.
  TORCH_INTERNAL_ASSERT_DEBUG_ONLY(infos.item().toInt() == 0);

  return self_working_copy;
}


void _cholesky_inverse_cusolver_potrs_based(Tensor& result, Tensor& infos, bool upper) {
  at::Tensor input_working_copy = cloneBatchedColumnMajor(result);
  at::Tensor infos_gpu = at::zeros({1}, result.options().dtype(at::kInt));
  result.fill_(0);
  result.diagonal(/*offset=*/0, /*dim1=*/-2, /*dim2=*/-1).fill_(1);
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(result.scalar_type(), "cholesky_cuda_potri", [&] {
    apply_cholesky_cusolver_potrs<scalar_t>(result, input_working_copy, upper, infos_gpu);
  });

  // Debug only: info of cusolver potrs only check if the i-th parameter is wrong
  // Function argument `infos` is a CPU tensor, the following copy will cause a device-host sync.
  // infos.copy_(infos_gpu);
}

Tensor& cholesky_inverse_kernel_impl_cusolver(Tensor &result, Tensor& infos, bool upper) {
  _cholesky_inverse_cusolver_potrs_based(result, infos, upper);
  return result;
}


/*
  The geqrf function computes the QR decomposition of a m x n matrix A.

  Args:
  * `A` - [in] Tensor with matrices for QR decomposition,
          [out] Tensor containing R in the upper triangle of A
          and elementary reflectors below the main diagonal of A
  * `tau` - Tensor containing the magnitudes of the elementary reflectors
  * `m` - The number of rows of `input` to consider
  * `n` - The number of columns of `input` to consider (actual sizes of `input` could be larger)

  For further details, please see the cuSOLVER documentation for GEQRF.
*/
template <typename scalar_t>
static void apply_geqrf(const Tensor& A, const Tensor& tau) {
  int64_t m = A.size(-2);
  int64_t n = A.size(-1);
  int64_t lda = std::max<int64_t>(1, m);
  int64_t batch_size = batchCount(A);

  auto A_stride = matrixStride(A);
  auto tau_stride = tau.size(-1);

  auto A_data = A.data_ptr<scalar_t>();
  auto tau_data = tau.data_ptr<scalar_t>();

  auto infos = at::zeros({1}, A.options().dtype(at::kInt));
  auto infos_data = infos.data_ptr<int>();

  // get the optimal work size and allocate workspace tensor
#ifdef USE_CUSOLVER_64_BIT
  size_t worksize_device; // workspaceInBytesOnDevice
  size_t worksize_host; // workspaceInBytesOnHost
  hipsolverDnParams_t params = NULL; // use default algorithm (currently it's the only option)
  at::cuda::solver::xgeqrf_bufferSize<scalar_t>(
      at::cuda::getCurrentCUDASolverDnHandle(),
      params,
      m,
      n,
      A_data,
      lda,
      tau_data,
      &worksize_device,
      &worksize_host);
#else
  int lwork;
  int m_32 = cuda_int_cast(m, "m");
  int n_32 = cuda_int_cast(n, "n");
  int lda_32 = cuda_int_cast(lda, "lda");
  at::cuda::solver::geqrf_bufferSize<scalar_t>(
      at::cuda::getCurrentCUDASolverDnHandle(), m_32, n_32, A_data, lda_32, &lwork);
#endif // USE_CUSOLVER_64_BIT

  for (decltype(batch_size) i = 0; i < batch_size; i++) {
    scalar_t* A_working_ptr = &A_data[i * A_stride];
    scalar_t* tau_working_ptr = &tau_data[i * tau_stride];
    auto handle = at::cuda::getCurrentCUDASolverDnHandle();

#ifdef USE_CUSOLVER_64_BIT
    // allocate workspace storage on device and host
    auto& device_allocator = *at::cuda::getCUDADeviceAllocator();
    auto work_device_data = device_allocator.allocate(worksize_device);
    auto& host_allocator = *at::getCPUAllocator();
    auto work_host_data = host_allocator.allocate(worksize_host);
    at::cuda::solver::xgeqrf<scalar_t>(
        handle,
        params,
        m,
        n,
        A_working_ptr,
        lda,
        tau_working_ptr,
        static_cast<scalar_t*>(work_device_data.get()),
        worksize_device,
        static_cast<scalar_t*>(work_host_data.get()),
        worksize_host,
        infos_data);
#else
    // allocate workspace storage on device
    auto& allocator = *at::cuda::getCUDADeviceAllocator();
    auto work_data = allocator.allocate(sizeof(scalar_t) * std::max<int>(1, lwork));
    at::cuda::solver::geqrf<scalar_t>(
        handle,
        m_32,
        n_32,
        A_working_ptr,
        lda_32,
        tau_working_ptr,
        static_cast<scalar_t*>(work_data.get()),
        lwork,
        infos_data);
#endif // USE_CUSOLVER_64_BIT
  }

  // info from geqrf only reports if the i-th parameter is wrong, not about the matrix singularity
  // so we don't need to check it all the time
  TORCH_INTERNAL_ASSERT_DEBUG_ONLY(infos.item().toInt() == 0);
}

// This is a type dispatching helper function for 'apply_geqrf'
void geqrf_cusolver(const Tensor& input, const Tensor& tau) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(input.scalar_type(), "geqrf_cuda", [&]{
    apply_geqrf<scalar_t>(input, tau);
  });
}

/*
  The ormqr function multiplies Q with another matrix from a sequence of
  elementary reflectors, such as is produced by the geqrf function.

  Args:
  * `input`     - Tensor with elementary reflectors below the diagonal,
                  encoding the matrix Q.
  * `tau`       - Tensor containing the magnitudes of the elementary
                  reflectors.
  * `other`     - [in] Tensor containing the matrix to be multiplied.
                  [out] result of the matrix multiplication with Q.
  * `left`      - bool, determining whether `other` is left- or right-multiplied with Q.
  * `transpose` - bool, determining whether to transpose (or conjugate transpose) Q before multiplying.

  For further details, please see the cuSOLVER documentation for ORMQR and UNMQR.
*/
template <typename scalar_t>
static void apply_ormqr(const Tensor& input, const Tensor& tau, const Tensor& other, bool left, bool transpose) {
  using value_t = typename c10::scalar_value_type<scalar_t>::type;

  auto side = left ? HIPBLAS_SIDE_LEFT : HIPBLAS_SIDE_RIGHT;
  auto trans = transpose ? (input.is_complex() ? HIPBLAS_OP_C : HIPBLAS_OP_T) : HIPBLAS_OP_N;

  auto input_data = input.data_ptr<scalar_t>();
  auto tau_data = tau.data_ptr<scalar_t>();
  auto other_data = other.data_ptr<scalar_t>();

  auto input_matrix_stride = matrixStride(input);
  auto other_matrix_stride = matrixStride(other);
  auto tau_stride = tau.size(-1);
  auto batch_size = batchCount(input);
  auto m = cuda_int_cast(other.size(-2), "m");
  auto n = cuda_int_cast(other.size(-1), "n");
  auto k = cuda_int_cast(tau.size(-1), "k");
  auto lda = std::max<int>(1, left ? m : n);
  auto ldc = std::max<int>(1, m);

  // get the optimal work size and allocate workspace tensor
  int lwork;
  at::cuda::solver::ormqr_bufferSize<scalar_t>(
    at::cuda::getCurrentCUDASolverDnHandle(), side, trans, m, n, k, input_data, lda, tau_data, other_data, ldc, &lwork);

  auto info = at::zeros({1}, input.options().dtype(at::kInt));
  auto info_data = info.data_ptr<int>();

  for (auto i = decltype(batch_size){0}; i < batch_size; i++) {
    scalar_t* input_working_ptr = &input_data[i * input_matrix_stride];
    scalar_t* other_working_ptr = &other_data[i * other_matrix_stride];
    scalar_t* tau_working_ptr = &tau_data[i * tau_stride];
    auto handle = at::cuda::getCurrentCUDASolverDnHandle();

    // allocate workspace storage
    auto& allocator = *at::cuda::getCUDADeviceAllocator();
    auto work_data = allocator.allocate(sizeof(scalar_t)*lwork);

    at::cuda::solver::ormqr<scalar_t>(
      handle, side, trans, m, n, k,
      input_working_ptr,
      lda,
      tau_working_ptr,
      other_working_ptr,
      ldc,
      static_cast<scalar_t*>(work_data.get()),
      lwork,
      info_data
    );

    // info from ormqr only reports if the i-th parameter is wrong
    // so we don't need to check it all the time
    TORCH_INTERNAL_ASSERT_DEBUG_ONLY(info.item().toInt() == 0);
  }
}

// This is a type dispatching helper function for 'apply_ormqr'
void ormqr_cusolver(const Tensor& input, const Tensor& tau, const Tensor& other, bool left, bool transpose) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(input.scalar_type(), "orgmr_cuda", [&]{
    apply_ormqr<scalar_t>(input, tau, other, left, transpose);
  });
}

/*
  The orgqr function allows reconstruction of an orthogonal (or unitary) matrix Q,
  from a sequence of elementary reflectors, such as produced by the geqrf function.

  Args:
  * `self` - Tensor with the directions of the elementary reflectors below the diagonal,
              it will be overwritten with the result
  * `tau` - Tensor containing the magnitudes of the elementary reflectors

  For further details, please see the cuSOLVER documentation for ORGQR and UNGQR.
*/
template <typename scalar_t>
inline static void apply_orgqr(Tensor& self, const Tensor& tau) {
  using value_t = typename c10::scalar_value_type<scalar_t>::type;
  auto self_data = self.data_ptr<scalar_t>();
  auto tau_data = tau.data_ptr<scalar_t>();
  auto self_matrix_stride = matrixStride(self);
  auto batchsize = cuda_int_cast(batchCount(self), "batch size");
  auto m = cuda_int_cast(self.size(-2), "m");
  auto n = cuda_int_cast(self.size(-1), "n");
  auto k = cuda_int_cast(tau.size(-1), "k");
  auto tau_stride = std::max<int>(1, k);
  auto lda = std::max<int>(1, m);

  // LAPACK's requirement
  TORCH_INTERNAL_ASSERT(m >= n);
  TORCH_INTERNAL_ASSERT(n >= k);

  // cuSOLVER doesn't compute anything for this case, which is wrong
  // the result should be a matrix with 1 on the diagonal
  if (k == 0) {
    self.fill_(0);
    self.diagonal(/*offset=*/0, /*dim1=*/-2, /*dim2=*/-1).fill_(1);
    return;
  }

  // get the optimal work size and allocate workspace tensor
  int lwork;
  at::cuda::solver::orgqr_buffersize<scalar_t>(
    at::cuda::getCurrentCUDASolverDnHandle(), m, n, k, self_data, lda, tau_data, &lwork);

  auto info = at::zeros({1}, self.options().dtype(at::kInt));
  auto info_data = info.data_ptr<int>();

  for (auto i = decltype(batchsize){0}; i < batchsize; i++) {
    scalar_t* self_working_ptr = &self_data[i * self_matrix_stride];
    scalar_t* tau_working_ptr = &tau_data[i * tau_stride];
    auto handle = at::cuda::getCurrentCUDASolverDnHandle();

    // allocate workspace storage
    auto& allocator = *at::cuda::getCUDADeviceAllocator();
    auto work_data = allocator.allocate(sizeof(scalar_t)*lwork);

    at::cuda::solver::orgqr<scalar_t>(
      handle, m, n, k,
      self_working_ptr,
      lda,
      tau_working_ptr,
      static_cast<scalar_t*>(work_data.get()),
      lwork,
      info_data
    );

    // info from orgqr only reports if the i-th parameter is wrong
    // so we don't need to check it all the time
    TORCH_INTERNAL_ASSERT_DEBUG_ONLY(info.item().toInt() == 0);
  }
}

// This is a type dispatching helper function for 'apply_orgqr'
Tensor& orgqr_helper_cusolver(Tensor& result, const Tensor& tau) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(result.scalar_type(), "orgqr_cuda", [&]{
    apply_orgqr<scalar_t>(result, tau);
  });
  return result;
}

template <typename scalar_t>
static void apply_syevd(Tensor& values, Tensor& vectors, Tensor& infos, bool upper, bool compute_eigenvectors) {
  using value_t = typename c10::scalar_value_type<scalar_t>::type;

  hipblasFillMode_t uplo = upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER;
  hipsolverEigMode_t jobz = compute_eigenvectors ? HIPSOLVER_EIG_MODE_VECTOR : HIPSOLVER_EIG_MODE_NOVECTOR;

  int64_t n = vectors.size(-1);
  int64_t lda = std::max<int64_t>(1, n);
  int64_t batch_size = batchCount(vectors);

  auto vectors_stride = matrixStride(vectors);
  auto values_stride = values.size(-1);

  auto vectors_data = vectors.data_ptr<scalar_t>();
  auto values_data = values.data_ptr<value_t>();
  auto infos_data = infos.data_ptr<int>();

  // get the optimal work size and allocate workspace tensor
#ifdef USE_CUSOLVER_64_BIT
  size_t worksize_device; // workspaceInBytesOnDevice
  size_t worksize_host; // workspaceInBytesOnHost
  hipsolverDnParams_t params = NULL; // use default algorithm (currently it's the only option)
  at::cuda::solver::xsyevd_bufferSize<scalar_t>(
      at::cuda::getCurrentCUDASolverDnHandle(),
      params,
      jobz,
      uplo,
      n,
      vectors_data,
      lda,
      values_data,
      &worksize_device,
      &worksize_host);
#else
  int lwork;
  int n_32 = cuda_int_cast(n, "n");
  int lda_32 = cuda_int_cast(lda, "lda");
  at::cuda::solver::syevd_bufferSize<scalar_t>(
      at::cuda::getCurrentCUDASolverDnHandle(), jobz, uplo, n_32, vectors_data, lda_32, values_data, &lwork);
#endif // USE_CUSOLVER_64_BIT

  for (decltype(batch_size) i = 0; i < batch_size; i++) {
    scalar_t* vectors_working_ptr = &vectors_data[i * vectors_stride];
    value_t* values_working_ptr = &values_data[i * values_stride];
    int* info_working_ptr = &infos_data[i];
    auto handle = at::cuda::getCurrentCUDASolverDnHandle();

#ifdef USE_CUSOLVER_64_BIT
    // allocate workspace storage on device and host
    auto& device_allocator = *at::cuda::getCUDADeviceAllocator();
    auto work_device_data = device_allocator.allocate(worksize_device);
    auto& host_allocator = *at::getCPUAllocator();
    auto work_host_data = host_allocator.allocate(worksize_host);
    at::cuda::solver::xsyevd<scalar_t>(
        handle,
        params,
        jobz,
        uplo,
        n,
        vectors_working_ptr,
        lda,
        values_working_ptr,
        static_cast<scalar_t*>(work_device_data.get()),
        worksize_device,
        static_cast<scalar_t*>(work_host_data.get()),
        worksize_host,
        info_working_ptr);
#else
    // allocate workspace storage on device
    auto& allocator = *at::cuda::getCUDADeviceAllocator();
    auto work_data = allocator.allocate(sizeof(scalar_t) * lwork);
    at::cuda::solver::syevd<scalar_t>(
        handle,
        jobz,
        uplo,
        n_32,
        vectors_working_ptr,
        lda_32,
        values_working_ptr,
        static_cast<scalar_t*>(work_data.get()),
        lwork,
        info_working_ptr);
#endif // USE_CUSOLVER_64_BIT
  }
}

template <typename scalar_t>
static void apply_syevj(Tensor& values, Tensor& vectors, Tensor& infos, bool upper, bool compute_eigenvectors) {
  using value_t = typename c10::scalar_value_type<scalar_t>::type;

  hipblasFillMode_t uplo = upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER;
  hipsolverEigMode_t jobz = compute_eigenvectors ? HIPSOLVER_EIG_MODE_VECTOR : HIPSOLVER_EIG_MODE_NOVECTOR;

  int n = cuda_int_cast(vectors.size(-1), "n");
  int lda = std::max<int>(1, n);
  auto batch_size = batchCount(vectors);

  auto vectors_stride = matrixStride(vectors);
  auto values_stride = values.size(-1);

  auto vectors_data = vectors.data_ptr<scalar_t>();
  auto values_data = values.data_ptr<value_t>();
  auto infos_data = infos.data_ptr<int>();

  // syevj_params controls the numerical accuracy of syevj
  // by default the tolerance is set to machine accuracy
  // the maximum number of iteration of Jacobi method by default is 100
  // cuSOLVER documentations says: "15 sweeps are good enough to converge to machine accuracy"
  // LAPACK has SVD routine based on similar Jacobi algorithm (gesvj) and there a maximum of 30 iterations is set
  // Let's use the default values for now
  hipsolverSyevjInfo_t syevj_params;
  TORCH_CUSOLVER_CHECK(hipsolverDnCreateSyevjInfo(&syevj_params));

  // get the optimal work size and allocate workspace tensor
  int lwork;
  at::cuda::solver::syevj_bufferSize<scalar_t>(
      at::cuda::getCurrentCUDASolverDnHandle(), jobz, uplo, n, vectors_data, lda, values_data, &lwork, syevj_params);

  for (decltype(batch_size) i = 0; i < batch_size; i++) {
    scalar_t* vectors_working_ptr = &vectors_data[i * vectors_stride];
    value_t* values_working_ptr = &values_data[i * values_stride];
    int* info_working_ptr = &infos_data[i];
    auto handle = at::cuda::getCurrentCUDASolverDnHandle();

    // allocate workspace storage on device
    auto& allocator = *at::cuda::getCUDADeviceAllocator();
    auto work_data = allocator.allocate(sizeof(scalar_t) * lwork);
    at::cuda::solver::syevj<scalar_t>(
        handle,
        jobz,
        uplo,
        n,
        vectors_working_ptr,
        lda,
        values_working_ptr,
        static_cast<scalar_t*>(work_data.get()),
        lwork,
        info_working_ptr,
        syevj_params);
  }
  TORCH_CUSOLVER_CHECK(hipsolverDnDestroySyevjInfo(syevj_params));
}

template <typename scalar_t>
static void apply_syevj_batched(Tensor& values, Tensor& vectors, Tensor& infos, bool upper, bool compute_eigenvectors) {
  using value_t = typename c10::scalar_value_type<scalar_t>::type;

  hipblasFillMode_t uplo = upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER;
  hipsolverEigMode_t jobz = compute_eigenvectors ? HIPSOLVER_EIG_MODE_VECTOR : HIPSOLVER_EIG_MODE_NOVECTOR;

  int n = cuda_int_cast(vectors.size(-1), "n");
  int lda = std::max<int>(1, n);
  int batch_size = cuda_int_cast(batchCount(vectors), "batch_size");

  auto vectors_data = vectors.data_ptr<scalar_t>();
  auto values_data = values.data_ptr<value_t>();
  auto infos_data = infos.data_ptr<int>();

  // syevj_params controls the numerical accuracy of syevj
  // by default the tolerance is set to machine accuracy
  // the maximum number of iteration of Jacobi method by default is 100
  // cuSOLVER documentations says: "15 sweeps are good enough to converge to machine accuracy"
  // LAPACK has SVD routine based on similar Jacobi algorithm (gesvj) and there a maximum of 30 iterations is set
  // Let's use the default values for now
  hipsolverSyevjInfo_t syevj_params;
  TORCH_CUSOLVER_CHECK(hipsolverDnCreateSyevjInfo(&syevj_params));
  TORCH_CUSOLVER_CHECK(hipsolverDnXsyevjSetSortEig(syevj_params, 1));

  auto handle = at::cuda::getCurrentCUDASolverDnHandle();

  // get the optimal work size and allocate workspace tensor
  int lwork;
  at::cuda::solver::syevjBatched_bufferSize<scalar_t>(
      handle,
      jobz,
      uplo,
      n,
      vectors_data,
      lda,
      values_data,
      &lwork,
      syevj_params,
      batch_size);

  // allocate workspace storage on device
  auto& allocator = *at::cuda::getCUDADeviceAllocator();
  auto work_data = allocator.allocate(sizeof(scalar_t) * lwork);
  at::cuda::solver::syevjBatched<scalar_t>(
      handle,
      jobz,
      uplo,
      n,
      vectors_data,
      lda,
      values_data,
      static_cast<scalar_t*>(work_data.get()),
      lwork,
      infos_data,
      syevj_params,
      batch_size);
  TORCH_CUSOLVER_CHECK(hipsolverDnDestroySyevjInfo(syevj_params));
}

static void linalg_eigh_cusolver_syevd(Tensor& eigenvalues, Tensor& eigenvectors, Tensor& infos, bool upper, bool compute_eigenvectors) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(eigenvectors.scalar_type(), "linalg_eigh_cuda", [&] {
    apply_syevd<scalar_t>(eigenvalues, eigenvectors, infos, upper, compute_eigenvectors);
  });
}

static void linalg_eigh_cusolver_syevj(Tensor& eigenvalues, Tensor& eigenvectors, Tensor& infos, bool upper, bool compute_eigenvectors) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(eigenvectors.scalar_type(), "linalg_eigh_cuda", [&] {
    apply_syevj<scalar_t>(eigenvalues, eigenvectors, infos, upper, compute_eigenvectors);
  });
}

void linalg_eigh_cusolver(Tensor& eigenvalues, Tensor& eigenvectors, Tensor& infos, bool upper, bool compute_eigenvectors) {
  // TODO: syevj_batched should be added here, but at least for CUDA 11.2 it contains a bug leading to incorrect results
  // See https://github.com/pytorch/pytorch/pull/53040#issuecomment-793626268 and https://github.com/cupy/cupy/issues/4847

  // syevj is better than syevd for float32 dtype and matrix sizes 32x32 - 512x512
  // See https://github.com/pytorch/pytorch/pull/53040#issuecomment-788264724
  if (eigenvectors.scalar_type() == at::kFloat && eigenvectors.size(-1) >= 32 && eigenvectors.size(-1) <= 512) {
    return linalg_eigh_cusolver_syevj(eigenvalues, eigenvectors, infos, upper, compute_eigenvectors);
  } else {
    return linalg_eigh_cusolver_syevd(eigenvalues, eigenvectors, infos, upper, compute_eigenvectors);
  }
}

#endif  // USE_CUSOLVER

}} // namespace at::native
