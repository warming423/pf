#include "hip/hip_runtime.h"
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/Dispatch.h>
#include <ATen/ExpandUtils.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>

namespace {

inline void lerp_cuda(at::Tensor& ret, const at::Tensor& self, const at::Tensor& end, const at::Tensor& weights) {
  TORCH_CHECK(self.dtype() == end.dtype(), "expected dtype ", self.dtype(), " for `end` but got dtype ", end.dtype());
  TORCH_CHECK(self.dtype() == weights.dtype(), "expected dtype ", self.dtype(), " for `weights` but got dtype ", weights.dtype());
  at::TensorIterator iter = at::TensorIteratorConfig()
      .add_output(ret)
      .add_input(self)
      .add_input(end)
      .add_input(weights)
      .build();
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16,
      iter.common_dtype(), "lerp_cuda",
      [&] {
        at::native::gpu_kernel(
            iter,
            [] GPU_LAMBDA(
                scalar_t self_val,
                scalar_t end_val,
                scalar_t weight_val) -> scalar_t {
              return (std::abs(weight_val) < 0.5)
                  ? self_val + weight_val * (end_val - self_val)
                  : end_val -
                      (end_val - self_val) *
                          (static_cast<scalar_t>(1) - weight_val);
            });
      });
}

inline void lerp_scalar_cuda(at::Tensor& ret, const at::Tensor& self, const at::Tensor& end, const c10::Scalar& weight) {
  TORCH_CHECK(self.dtype() == end.dtype(), "expected dtype ", self.dtype(), " for `end` but got dtype ", end.dtype());
  at::TensorIterator iter = at::TensorIteratorConfig()
      .add_output(ret)
      .add_input(self)
      .add_input(end)
      .build();
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16,
      self.scalar_type(), "lerp_cuda",
      [&]{
        auto weight_val = weight.to<scalar_t>();
        at::native::gpu_kernel(
            iter, [=] GPU_LAMBDA(scalar_t self_val, scalar_t end_val) {
              return (std::abs(weight_val) < 0.5)
                  ? self_val + weight_val * (end_val - self_val)
                  : end_val -
                      (end_val - self_val) * (static_cast<scalar_t>(1) - weight_val);
            });
      });
    }
} // anonymous namespace

namespace at {
namespace native {

Tensor& lerp_cuda_tensor_out(const Tensor& self,
                            const Tensor& end, const Tensor& weight, Tensor& result) {
  c10::MaybeOwned<Tensor> b_self, b_end, b_weight;
  std::tie(b_self, b_end, b_weight) = expand_outplace(self, end, weight, "lerp_out_cuda");
  lerp_cuda(result, *b_self, *b_end, *b_weight);
  return result;
}

Tensor& lerp_cuda_scalar_out(const Tensor& self,
                            const Tensor& end, const Scalar& weight, Tensor& result) {
  c10::MaybeOwned<Tensor> b_self, b_end;
  std::tie(b_self, b_end) = expand_outplace(self, end, "lerp_out_cuda");
  lerp_scalar_cuda(result, *b_self, *b_end, weight);
  return result;
}

Tensor& lerp_cuda_tensor_(Tensor& self, const Tensor& end, const Tensor& weight) {
  c10::MaybeOwned<Tensor> b_self, b_end, b_weight;
  std::tie(b_self, b_end, b_weight) = expand_outplace(self, end, weight, "lerp__cuda");
  TORCH_CHECK(b_self->sizes() == self.sizes(),
           "output with shape ", self.sizes(),
           " doesn't match the broadcast shape ", b_self->sizes());
  lerp_cuda(self, *b_self, *b_end, *b_weight);
  return self;
}

Tensor& lerp_cuda_scalar_(Tensor& self, const Tensor& end, const Scalar& weight) {
  c10::MaybeOwned<Tensor> b_self, b_end;
  std::tie(b_self, b_end) = expand_outplace(self, end, "lerp__cuda");
  TORCH_CHECK(b_self->sizes() == self.sizes(),
           "output with shape ", self.sizes(),
           " doesn't match the broadcast shape ", b_self->sizes());
  lerp_scalar_cuda(self, *b_self, *b_end, weight);
  return self;
}

Tensor lerp_cuda_tensor(const Tensor& self, const Tensor& end, const Tensor& weight) {
  c10::MaybeOwned<Tensor> b_self, b_end, b_weight;
  std::tie(b_self, b_end, b_weight) = expand_outplace(self, end, weight, "lerp_cuda");
  Tensor result = at::empty_like(*b_self, b_self->suggest_memory_format());
  lerp_cuda(result, *b_self, *b_end, *b_weight);
  return result;
}

Tensor lerp_cuda_scalar(const Tensor& self, const Tensor& end, const Scalar& weight) {
  c10::MaybeOwned<Tensor> b_self, b_end;
  std::tie(b_self, b_end) = expand_outplace(self, end, "lerp_cuda");
  Tensor result = at::empty_like(*b_self, b_self->suggest_memory_format());
  lerp_scalar_cuda(result, *b_self, *b_end, weight);
  return result;
}

} // namespace native
} // namespace at
