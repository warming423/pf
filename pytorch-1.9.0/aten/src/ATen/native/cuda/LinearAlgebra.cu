#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/LegacyTHFunctionsCUDA.h>
#include <ATen/NamedTensorUtils.h>
#include <ATen/cuda/CUDABlas.h>
#include <ATen/Dispatch.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/LinearAlgebra.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/Resize.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/Reduce.cuh>
#include <ATen/native/SharedReduceOps.h>
#include <ATen/native/ReduceOps.h>
#include <c10/util/MaybeOwned.h>

namespace at { namespace native {

namespace {

c10::MaybeOwned<Tensor> inline prepare_matrix_for_cublas(const Tensor& tensor, bool& transpose_tensor) {
  if (tensor.is_non_overlapping_and_dense()) { // common case
      transpose_tensor = tensor.is_contiguous();
      return c10::MaybeOwned<Tensor>::borrowed(tensor);
  }
  IntArrayRef tensor_strides = tensor.strides();
  IntArrayRef tensor_sizes = tensor.sizes();
  if ((tensor_strides[0] == 1) && (tensor_strides[1] >= std::max<int64_t>(1, tensor_sizes[0]))) {
    transpose_tensor = false;
    return c10::MaybeOwned<Tensor>::borrowed(tensor);
  } else if ((tensor_strides[1] == 1) && (tensor_strides[0] >= std::max<int64_t>(1, tensor_sizes[1]))) {
    transpose_tensor = true;
    return c10::MaybeOwned<Tensor>::borrowed(tensor);
  } else {
    transpose_tensor = true;
    return c10::MaybeOwned<Tensor>::owned(tensor.clone(at::MemoryFormat::Contiguous));
  }
}

} // namespace

c10::MaybeOwned<Tensor> prepare_batch_matrix_for_cublas(const Tensor& tensor, bool& transpose_tensor, int64_t& ld_tensor, bool transpose_result, int64_t m, int64_t n) {
  IntArrayRef tensor_strides = tensor.strides();
  c10::MaybeOwned<Tensor> tensor_;
  int fast_dim = transpose_result ? 2 : 1;
  int leading_dim = transpose_result ? 1 : 2;

  if (tensor_strides[fast_dim] == 1 &&
    (tensor_strides[leading_dim] >= std::max<int64_t>(1, m))) {
    transpose_tensor = false;
    tensor_ = c10::MaybeOwned<Tensor>::borrowed(tensor);
    ld_tensor = tensor_strides[leading_dim];
  } else if ((tensor_strides[leading_dim] == 1) &&
    (tensor_strides[fast_dim] >= std::max<int64_t>(1, n))) {
    transpose_tensor = true;
    tensor_ = c10::MaybeOwned<Tensor>::borrowed(tensor);
    ld_tensor = tensor_strides[fast_dim];
  } else {
    transpose_tensor = !transpose_result;
    // gemm call requires leading dimension and stride parameters to be non-zero
    bool is_stride_non_zero = tensor.strides()[1] != 0 && tensor.strides()[2] != 0;
    if (tensor.is_contiguous() && is_stride_non_zero) {
      tensor_ = c10::MaybeOwned<Tensor>::borrowed(tensor);
    } else {
      tensor_ = c10::MaybeOwned<Tensor>::owned(tensor.clone(at::MemoryFormat::Contiguous));
    }
    ld_tensor = tensor_->strides()[1];
  }

  return tensor_;
}

namespace {

Tensor& addmm_out_cuda_impl(Tensor& result, const Tensor& self, const Tensor& mat1, const Tensor& mat2, const Scalar& beta, const Scalar& alpha) {
  // Make sure to keep addmm_cuda below in sync with this code; it
  // preflights a check to try to avoid actually needing to call
  // expand().
  TORCH_CHECK(mat1.dim() == 2 && mat2.dim() == 2, "tensors must be 2-D");

  TensorArg args[]{{result, "out", 0}, {self, "self", 1}, {mat1, "mat1", 2}, {mat2, "mat2", 3}};
  checkAllSameGPU("addmm", args);

  IntArrayRef mat1_sizes = mat1.sizes();
  IntArrayRef mat2_sizes = mat2.sizes();
  IntArrayRef self__sizes;
  c10::MaybeOwned<Tensor> self_;
  if (&result != &self) {
    self_ = expand_size(self, {mat1_sizes[0], mat2_sizes[1]}, "addmm");
    self__sizes = self_->sizes();
  } else {
    self_ = c10::MaybeOwned<Tensor>::borrowed(self);
    self__sizes = self_->sizes();
    TORCH_CHECK(result.dim() == 2, "tensors must be 2-D");
    TORCH_CHECK(self__sizes[0] == mat1_sizes[0], "self_ dim 0 must match mat1 dim 0");
    TORCH_CHECK(self__sizes[1] == mat2_sizes[1], "self_ dim 1 must match mat2 dim 1");
  }

  if (&result != &self) {
    at::native::resize_output(result, self__sizes);
    if (beta.toComplexDouble() != 0.0) {
      at::native::copy_(result, *self_);
    }
  }


  IntArrayRef result_sizes = result.sizes();
  if ((result_sizes[0] == 0) || (result_sizes[1] == 0)) {
    return result;
  }

  bool transpose_result;
  c10::MaybeOwned<Tensor> result_ = prepare_matrix_for_cublas(result, transpose_result);
  bool transpose_mat1;
  bool transpose_mat2;
  c10::MaybeOwned<Tensor> mat1_ = prepare_matrix_for_cublas(transpose_result ? mat2 : mat1, transpose_mat1);
  c10::MaybeOwned<Tensor> mat2_ = prepare_matrix_for_cublas(transpose_result ? mat1 : mat2, transpose_mat2);

  if (transpose_result) {
    transpose_mat1 = !transpose_mat1;
    transpose_mat2 = !transpose_mat2;
    mat1_sizes = mat1_->sizes();
    mat2_sizes = mat2_->sizes();
  }

  int64_t m = mat1_sizes[transpose_result ? 1 : 0];
  int64_t k = mat1_sizes[transpose_result ? 0 : 1];
  int64_t n = mat2_sizes[transpose_result ? 0 : 1];
  int64_t mat1_ld = mat1_->stride((transpose_mat1 == transpose_result) ? 1 : 0);
  int64_t mat2_ld = mat2_->stride((transpose_mat2 == transpose_result) ? 1 : 0);
  int64_t result_ld = result_->stride(transpose_result ? 0 : 1);
  at::ScalarType scalar_type = self_->scalar_type();

  if (mat1.numel() == 0) {
    // By definition, when beta==0, values in self should be ignored. nans and infs
    // should not propagate
    if (beta.toComplexDouble() == 0.) {
      return result.zero_();
    }
    // TODO: We could squeeze some perf by calling at::cuda::mul_out here instead, to bypass the dispatcher.
    // That requires some fixing some internal build dependencies though.
    return at::mul_out(
        result,
        self,
        at::native::scalar_tensor(
            beta,
            self.scalar_type(),
            c10::nullopt /* layout */,
            at::kCPU,
            c10::nullopt /* pin_memory */));
  }

  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, scalar_type, "addmm_cuda", [&] {
    scalar_t alpha_val = alpha.to<scalar_t>();
    scalar_t beta_val = beta.to<scalar_t>();
    scalar_t* mat1_ptr = mat1_->data_ptr<scalar_t>();
    scalar_t* mat2_ptr = mat2_->data_ptr<scalar_t>();
    scalar_t* result_ptr = result_->data_ptr<scalar_t>();
    at::cuda::blas::gemm<scalar_t>(
      transpose_mat1 ? 't' : 'n',
      transpose_mat2 ? 't' : 'n',
      m, n, k,
      alpha_val,
      mat1_ptr, mat1_ld,
      mat2_ptr, mat2_ld,
      beta_val,
      result_ptr, result_ld
    );
  });
  if (!result.is_same(*result_)) {
    result.copy_(*result_);
  }
  return result;
}

Tensor& baddbmm_out_cuda_impl(Tensor& result, const Tensor& self, const Tensor& batch1, const Tensor& batch2, const Scalar& beta, const Scalar& alpha) {
  TORCH_CHECK(self.dim() == 3, "self must be a 3D tensor");
  TORCH_CHECK(batch1.dim() == 3, "batch1 must be a 3D tensor");
  TORCH_CHECK(batch2.dim() == 3, "batch2 must be a 3D tensor");

  TensorArg args[]{{result, "out", 0}, {self, "self", 1}, {batch1, "batch1", 2}, {batch2, "batch2", 3}};
  checkAllSameGPU("baddbmm", args);

  IntArrayRef batch1_sizes = batch1.sizes();
  IntArrayRef batch2_sizes = batch2.sizes();
  IntArrayRef self_sizes = self.sizes();

  TORCH_CHECK(self_sizes[0] == batch1_sizes[0], "self dim 0 must match batch1 dim 0");
  TORCH_CHECK(self_sizes[0] == batch2_sizes[0], "self dim 0 must match batch2 dim 0");
  TORCH_CHECK(self_sizes[1] == batch1_sizes[1], "self dim 1 must match batch1 dim 1");
  TORCH_CHECK(self_sizes[2] == batch2_sizes[2], "self dim 2 must match batch2 dim 2");
  TORCH_CHECK(batch1_sizes[2] == batch2_sizes[1], "batch1 dim 2 must match batch2 dim 1");

  if (!result.is_same(self)) {
    result.resize_as_(self);
    if (beta.to<c10::complex<double>>() != 0.0) {
      result.copy_(self);
    }
  }

  // handle pathological cases that blas may not like
  if (result.numel() == 0) {
    return result;
  } else if (batch1_sizes[2] == 0) {
    if (beta.to<c10::complex<double>>() == 0.0) {
      return result.zero_();
    } else {
      return result.mul_(beta);
    }
  }

  bool transpose_result = false;
  c10::MaybeOwned<Tensor> result_;
  IntArrayRef result_strides = result.strides();
  IntArrayRef result_sizes = result.sizes();

  if ((result_strides[1] == 1) &&
      ((result_sizes[2] == 1) || (result_strides[2] >= std::max<int64_t>(1, result_sizes[1])))) {
    result_ = c10::MaybeOwned<Tensor>::borrowed(result);
  } else if ((result_strides[2] == 1) &&
    (result_sizes[1] == 1 || (result_strides[1] >= std::max<int64_t>(1, result_sizes[2])))) {
    transpose_result = true;
    result_ = c10::MaybeOwned<Tensor>::borrowed(result);
  } else {
    result_ = c10::MaybeOwned<Tensor>::owned(result.transpose(1, 2).clone(at::MemoryFormat::Contiguous).transpose(1, 2));
  }

  int leading_dim = transpose_result ? 1 : 2;

  int64_t m = result_sizes[transpose_result ? 2 : 1];
  int64_t n = result_sizes[leading_dim];
  int64_t k = (transpose_result ? batch2 : batch1).sizes()[leading_dim];

  int64_t lda, ldb, ldc;
  bool transpose_batch1, transpose_batch2;
  auto batch1_ = prepare_batch_matrix_for_cublas(transpose_result ? batch2 : batch1, transpose_batch1, lda, transpose_result, m, k);
  auto batch2_ = prepare_batch_matrix_for_cublas(transpose_result ? batch1 : batch2, transpose_batch2, ldb, transpose_result, k, n);

  ldc = result_->strides()[leading_dim];
  int64_t num_batches = result_->sizes()[0];

  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, self.scalar_type(), "baddbmm_cuda", [&] {
    scalar_t alpha_val = alpha.to<scalar_t>();
    scalar_t beta_val = beta.to<scalar_t>();
    scalar_t* batch1_ptr = batch1_->data_ptr<scalar_t>();
    scalar_t* batch2_ptr = batch2_->data_ptr<scalar_t>();
    scalar_t* result_ptr = result_->data_ptr<scalar_t>();
    at::cuda::blas::bgemm<scalar_t>(
      transpose_batch1 ? 't' : 'n',
      transpose_batch2 ? 't' : 'n',
      m, n, k,
      alpha_val,
      batch1_ptr, lda, batch1_->strides()[0],
      batch2_ptr, ldb, batch2_->strides()[0],
      beta_val,
      result_ptr, ldc, result_->strides()[0],
      num_batches
    );
  });
  if (!result.is_same(*result_)) {
    result.copy_(*result_);
  }
  return result;
}

} // anonymous namespace

TORCH_IMPL_FUNC(addmm_out_cuda)(const Tensor& self, const Tensor& mat1, const Tensor& mat2, const Scalar& beta, const Scalar& alpha, const Tensor& result) {
  addmm_out_cuda_impl(const_cast<Tensor&>(result), self, mat1, mat2, beta, alpha);
}

Tensor& mm_out_cuda(const Tensor& self, const Tensor& mat2, Tensor& result) {
  result.resize_({ self.size(0), mat2.size(1) });
  return addmm_out_cuda_impl(result, result, self, mat2, 0, 1);
}

Tensor mm_cuda(const Tensor& self, const Tensor& mat2) {
  Tensor result = at::empty({ self.size(0), mat2.size(1) }, self.options());
  return addmm_out_cuda_impl(result, result, self, mat2, 0, 1);
}

Tensor& baddbmm_out_cuda(const Tensor& self, const Tensor& batch1, const Tensor& batch2, const Scalar& beta, const Scalar& alpha, Tensor &result) {
  auto self_ = &result == &self
    ? c10::MaybeOwned<Tensor>::borrowed(self)
    : expand_size(self, {batch1.size(0), batch1.size(1), batch2.size(2)}, "baddbmm");
  {
    at::NoNamesGuard guard;
    baddbmm_out_cuda_impl(result, *self_, batch1, batch2, beta, alpha);
  }
  namedinference::propagate_names_if_nonempty(
       result,
       namedinference::compute_baddbmm_outnames(result, batch1, batch2, self));
  return result;
}

Tensor baddbmm_cuda(const Tensor& self, const Tensor& batch1, const Tensor& batch2, const Scalar& beta, const Scalar& alpha) {
  Tensor out = at::empty({0}, self.options());
  return baddbmm_out_cuda(self, batch1, batch2, beta, alpha, out);
}

Tensor& baddbmm__cuda(Tensor& self, const Tensor& batch1, const Tensor& batch2, const Scalar& beta, const Scalar& alpha) {
  return baddbmm_out_cuda(self, batch1, batch2, beta, alpha, self);
}

Tensor& bmm_out_cuda(const Tensor& batch1, const Tensor& batch2, Tensor &result) {
  TORCH_CHECK(batch1.dim() == 3, "batch1 must be a 3D tensor");
  TORCH_CHECK(batch2.dim() == 3, "batch2 must be a 3D tensor");
  at::native::resize_output(result, {batch1.sizes()[0], batch1.sizes()[1], batch2.sizes()[2]});
  Scalar beta(0.0);
  Scalar alpha(1.0);
  {
    NoNamesGuard guard;
    baddbmm_out_cuda_impl(result, result, batch1, batch2, beta, alpha);
  }
  namedinference::propagate_names_if_nonempty(
      result,
      namedinference::compute_bmm_outnames(result, batch1, batch2));
  return result;
}

Tensor bmm_cuda(const Tensor& self, const Tensor& mat2) {
  TORCH_CHECK(self.dim() == 3, "self must be a 3D tensor");
  TORCH_CHECK(mat2.dim() == 3, "batch2 must be a 3D tensor");
  Tensor result = at::empty({self.sizes()[0], self.sizes()[1], mat2.sizes()[2]}, self.options());
  return native::bmm_out_cuda(self, mat2, result);
}

namespace {

inline void dot_check(const Tensor& self, const Tensor& other) {
  TORCH_CHECK(
      self.dim() == 1 && other.dim() == 1,
      "1D tensors expected, but got ",
      self.dim(),
      "D and ",
      other.dim(),
      "D tensors");
  TORCH_CHECK(
      self.scalar_type() == other.scalar_type(),
      "dot : expected both vectors to have same dtype, but found ",
      self.scalar_type(),
      " and ",
      other.scalar_type());
  TORCH_CHECK(
      self.numel() == other.numel(),
      "inconsistent tensor size, expected tensor [",
      self.numel(),
      "] and src [",
      other.numel(),
      "] to have the same number of elements, but got ",
      self.numel(),
      " and ",
      other.numel(),
      " elements respectively");
  TORCH_CHECK(
      self.device() == other.device(),
      "expected all tensors to be on the same device. Found: ",
      self.device(),
      ", ",
      other.device());
  TORCH_CHECK(
      (self.numel() <= INT_MAX) && (self.stride(0) <= INT_MAX) &&
          (other.stride(0) <= INT_MAX),
      "dot only supports n, incx, incy with the bound [val] <= %d",
      INT_MAX);
}

} // anonymous namespace

Tensor dot_cuda(const Tensor& self, const Tensor& other) {
  at::NoNamesGuard guard;

  dot_check(self, other);

  const int n = static_cast<int>(self.numel());
  int incx = static_cast<int>(self.stride(0));
  int incy = static_cast<int>(other.stride(0));
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

return AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(
      ScalarType::Half, ScalarType::BFloat16,
      self.scalar_type(), "dot",
      [&] {
        Tensor result = at::empty({}, self.options());

        auto handle = at::cuda::getCurrentCUDABlasHandle();
        at::cuda::blas::PointerModeGuard pointerModeGuard(handle, HIPBLAS_POINTER_MODE_DEVICE);
        at::cuda::blas::dot<scalar_t>(
            handle,
            n,
            self.data_ptr<scalar_t>(),
            incx,
            other.data_ptr<scalar_t>(),
            incy,
            result.data_ptr<scalar_t>());

        return result;
      });
}

Tensor vdot_cuda(const Tensor& self, const Tensor& other) {
  if (!self.is_complex()) {
    return dot_cuda(self, other);
  }

  at::NoNamesGuard guard;
  dot_check(self, other);

  const int n = static_cast<int>(self.numel());
  int incx = static_cast<int>(self.stride(0));
  int incy = static_cast<int>(other.stride(0));
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  return AT_DISPATCH_COMPLEX_TYPES(self.scalar_type(), "vdot", [&] {
    Tensor result = at::empty({}, self.options());

    auto handle = at::cuda::getCurrentCUDABlasHandle();
    at::cuda::blas::PointerModeGuard pointerModeGuard(
        handle, HIPBLAS_POINTER_MODE_DEVICE);
    at::cuda::blas::vdot<scalar_t>(
        handle,
        n,
        self.data_ptr<scalar_t>(),
        incx,
        other.data_ptr<scalar_t>(),
        incy,
        result.data_ptr<scalar_t>());

    return result;
  });
}

namespace {

void addr_kernel_cuda(TensorIterator &iter, const Scalar& beta, const Scalar& alpha) {
  if (iter.dtype() == ScalarType::Bool) {
    using scalar_t = bool;
    auto beta_val = beta.to<scalar_t>();
    auto alpha_val = alpha.to<scalar_t>();

    // when beta is false, values in self should be ignored,
    // nans and infs in self should not propagate.
    if (beta_val == false) {
      gpu_kernel(
        iter,
        [=] GPU_LAMBDA (scalar_t self_val,
                        scalar_t vec1_val, scalar_t vec2_val) -> scalar_t {
          return alpha_val && vec1_val && vec2_val;
        }
      );
    } else {
      gpu_kernel(
        iter,
        [=] GPU_LAMBDA (scalar_t self_val,
                        scalar_t vec1_val, scalar_t vec2_val) -> scalar_t {
          return (beta_val && self_val) || (alpha_val && vec1_val && vec2_val);
        }
      );
    }
    return;
  }

  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(kBFloat16, kHalf,
                                         iter.dtype(), "addr_cuda", [&] {
    auto beta_val = beta.to<scalar_t>();
    auto alpha_val = alpha.to<scalar_t>();

    scalar_t zero_val(0);
    // when beta==0, values in self should be ignored,
    // nans and infs in self should not propagate.
    if (beta_val == zero_val) {
      gpu_kernel(
        iter,
        [=] GPU_LAMBDA (scalar_t self_val,
                        scalar_t vec1_val, scalar_t vec2_val) -> scalar_t {
          return alpha_val * vec1_val * vec2_val;
        }
      );
    } else {
      gpu_kernel(
        iter,
        [=] GPU_LAMBDA (scalar_t self_val,
                        scalar_t vec1_val, scalar_t vec2_val) -> scalar_t {
          return beta_val * self_val + alpha_val * vec1_val * vec2_val;
        }
      );
    }
  });
}


template <int n_threads, int n_elems_per_thread, typename func_t>
C10_LAUNCH_BOUNDS_2(n_threads, n_elems_per_thread)
__global__ void _elementwise_kernel(int total_n_elems, func_t f) {
  constexpr int total_work_block = n_threads * n_elems_per_thread;
  int idx = total_work_block * blockIdx.x + threadIdx.x;

  #pragma unroll
  for (int i = 0; i < n_elems_per_thread; ++i) {
    if (idx < total_n_elems) {
      f(idx);
      idx += n_threads;
    }
  }
}

template <int n_threads, int n_elems_per_thread, typename func_t>
static void _launch_kernel(int total_n_elems, func_t f) {
  TORCH_INTERNAL_ASSERT(
    total_n_elems >= 0 && total_n_elems <= std::numeric_limits<int32_t>::max()
  );

  dim3 block(n_threads);
  constexpr int total_work_block = n_threads * n_elems_per_thread;
  dim3 grid((total_n_elems + total_work_block - 1) / total_work_block);

  auto stream = at::cuda::getCurrentCUDAStream();
  _elementwise_kernel<n_threads, n_elems_per_thread, func_t>
    <<<grid, block, 0, stream>>>(total_n_elems, f);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

void _unpack_pivots_internal_kernel(
  TensorIterator& iter,
  int64_t dim_size
) {
  if (iter.numel() == 0) {
    return;
  }

  if (!iter.can_use_32bit_indexing()) {
    for (auto& sub_iter : iter.with_32bit_indexing()) {
      _unpack_pivots_internal_kernel(sub_iter, dim_size);
    }
    return;
  }

  auto offset_calculator = make_offset_calculator<2>(iter);

  char* unpacked_pivots_ptr = reinterpret_cast<char*>(iter.data_ptr(0));
  const char* const __restrict__ pivots_ptr = reinterpret_cast<const char*>(iter.data_ptr(1));

  auto loop = [=]C10_DEVICE(int i) {
    auto offsets = offset_calculator.get(i);

    auto* unpacked_pivots_data = reinterpret_cast<int32_t*>(
      unpacked_pivots_ptr + offsets[0]);
    const auto* const __restrict__ pivots_data = reinterpret_cast<const int32_t*>(
      pivots_ptr + offsets[1]);

    // QUESTION: can we mix 64bit offsets with 32bit Iterator indexing?
    for (int64_t i = 0; i < dim_size; ++i) {
      thrust::swap(
        unpacked_pivots_data[i],
        unpacked_pivots_data[pivots_data[i]]
      );
    }
  };

  _launch_kernel<num_threads, thread_work_size>(iter.numel(), loop);
}

void unpack_pivots_cuda_kernel(
  TensorIterator& iter,
  int64_t dim_size
) {
  _unpack_pivots_internal_kernel(iter, dim_size);
}

} // anonymous namespace

REGISTER_DISPATCH(addr_stub, &addr_kernel_cuda);
REGISTER_DISPATCH(unpack_pivots_stub, &unpack_pivots_cuda_kernel);

}}
